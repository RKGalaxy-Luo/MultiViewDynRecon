#include "hip/hip_runtime.h"
/*****************************************************************//**
 * \file   ImageGradient.cu
 * \brief  ����ͼ����ݶȣ������࣬���ṩ�������ṩ�ڴ�
 * 
 * \author LUO
 * \date   March 24th 2024
 *********************************************************************/
#include "ImageGradient.h"

//			   | -3   0   3 |					| -3   -10   -3	|
// dx = 1/16 * | -10  0	 10 |		dy = 1/16 * | 0     0     0	|
//             | -3   0   3 |					| 3     10    3	|

__device__ void SparseSurfelFusion::device::computeImageGradient(const float v[8], float& dv_dx, float& dv_dy)
{
	dv_dx = 0.0625f * (-3 * v[0] + 3 * v[5] - 10 * v[1] + 10 * v[6] - 3 * v[2] + 3 * v[7]);
	dv_dy = 0.0625f * (-3 * v[0] + 3 * v[2] - 10 * v[3] + 10 * v[4] - 3 * v[5] + 3 * v[7]);
}

__global__ void SparseSurfelFusion::device::computeDensityForegroundMaskGradientKernel(hipTextureObject_t foregroundMask, hipTextureObject_t GrayscaleMap, unsigned int rows, unsigned int cols, hipSurfaceObject_t foregroundMaskGradientMap, hipSurfaceObject_t GrayscaleGradientMap)
{
	const unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
	const unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
	if (x >= cols || y >= rows) return;

	//�����ݶȱ���д��������
	float map_value[8];
	float2 mask_gradient, density_gradient;

	//���ز�����
	map_value[0] = tex2D<float>(foregroundMask, x - 1, y - 1);
	map_value[1] = tex2D<float>(foregroundMask, x - 1, y);
	map_value[2] = tex2D<float>(foregroundMask, x - 1, y + 1);
	map_value[3] = tex2D<float>(foregroundMask, x, y - 1);
	map_value[4] = tex2D<float>(foregroundMask, x, y + 1);
	map_value[5] = tex2D<float>(foregroundMask, x + 1, y - 1);
	map_value[6] = tex2D<float>(foregroundMask, x + 1, y);
	map_value[7] = tex2D<float>(foregroundMask, x + 1, y + 1);
	computeImageGradient(map_value, mask_gradient.x, mask_gradient.y);

	map_value[0] = tex2D<float>(GrayscaleMap, x - 1, y - 1);
	map_value[1] = tex2D<float>(GrayscaleMap, x - 1, y);
	map_value[2] = tex2D<float>(GrayscaleMap, x - 1, y + 1);
	map_value[3] = tex2D<float>(GrayscaleMap, x, y - 1);
	map_value[4] = tex2D<float>(GrayscaleMap, x, y + 1);
	map_value[5] = tex2D<float>(GrayscaleMap, x + 1, y - 1);
	map_value[6] = tex2D<float>(GrayscaleMap, x + 1, y);
	map_value[7] = tex2D<float>(GrayscaleMap, x + 1, y + 1);
	computeImageGradient(map_value, density_gradient.x, density_gradient.y);

	//��ֵ�洢��surface
	surf2Dwrite(mask_gradient, foregroundMaskGradientMap, x * sizeof(float2), y);
	surf2Dwrite(density_gradient, GrayscaleGradientMap, x * sizeof(float2), y);
}

void SparseSurfelFusion::ImageGradient::computeDensityForegroundMaskGradient(hipTextureObject_t filteredForegroundMask, hipTextureObject_t GrayscaleMap, unsigned int rows, unsigned int cols, hipSurfaceObject_t foregroundMaskGradientMap, hipSurfaceObject_t GrayscaleGradientMap, hipStream_t stream)
{
	dim3 block(16, 16);
	dim3 grid(divUp(cols, block.x), divUp(rows, block.y));
	device::computeDensityForegroundMaskGradientKernel << <grid, block, 0, stream >> > (filteredForegroundMask, GrayscaleMap, rows, cols, foregroundMaskGradientMap, GrayscaleGradientMap);
}
