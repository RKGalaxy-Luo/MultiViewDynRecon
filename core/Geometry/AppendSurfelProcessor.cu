#include "hip/hip_runtime.h"
#include <math/DualQuaternion/DualQuaternion.h>
#include <math/VectorUtils.h>
#include "append_surfel_collision.cuh"
#include "AppendSurfelProcessor.h"
#include "KNNSearch.h"
#include "KNNBruteForceLiveNodes.h"
#include <>

namespace SparseSurfelFusion { 
	namespace device {
		//The kernel to build the candidate surfel and finite diff vertex
		__global__ void buildCandidateSurfelAndFiniteDiffVertexKernel(
			AppendSurfelInput input,
			DeviceArrayView<ushort4> candidate_pixel,
			const float finitediff_step,
			//The output
			float4* finitediff_vertex,
			float4* surfel_vertex_confid,
			float4* surfel_normal_radius,
			float4* surfel_color_time
		) {
			const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
			if (idx < candidate_pixel.Size()) {
				const ushort4 pixel = candidate_pixel[idx];
				unsigned short CameraID = pixel.z;
				unsigned short isInterpolatedSurfels = pixel.w;
				float4 depth_vertex_confid, depth_normal_radius, color_time;
				if (isInterpolatedSurfels == 0) {	// ֻ���Լ��ӽ��µĵ����任
					depth_vertex_confid = tex2D<float4>(input.vertex_confid_map[CameraID], pixel.x, pixel.y);
					depth_normal_radius = tex2D<float4>(input.normal_radius_map[CameraID], pixel.x, pixel.y);
					color_time = tex2D<float4>(input.color_time_map[CameraID], pixel.x, pixel.y);
				}
				else {
					depth_vertex_confid = input.inter_vertex_map[CameraID](pixel.y, pixel.x);
					depth_normal_radius = input.inter_normal_map[CameraID](pixel.y, pixel.x);
					color_time = input.inter_color_map[CameraID](pixel.y, pixel.x);
				}

				// �Ƚ�ԭʼ����ת��0������ϵ��
				float3 depth_vertex_confid_0 = input.InitialCameraSE3[CameraID].rot * depth_vertex_confid + input.InitialCameraSE3[CameraID].trans;
				float3 depth_normal_radius_0 = input.InitialCameraSE3[CameraID].rot * depth_normal_radius;

				// ����ת��Live����
				float3 vertex = input.m_camera2world[CameraID].rot * depth_vertex_confid_0 + input.m_camera2world[CameraID].trans;	// �����ӽǵĵ�ת��Live����
				float3 normal = input.m_camera2world[CameraID].rot * depth_normal_radius_0;

				// ����ȫ�ǲ�ͬ�ӽǵĵ���Live���е�λ��
				const unsigned int offset = idx * 4;
				finitediff_vertex[offset + 0] = make_float4(vertex.x, vertex.y, vertex.z, depth_vertex_confid.w); // Note that the last element is confidence
				finitediff_vertex[offset + 1] = make_float4(vertex.x + finitediff_step, vertex.y, vertex.z, 1.0f);
				finitediff_vertex[offset + 2] = make_float4(vertex.x, vertex.y + finitediff_step, vertex.z, 1.0f);
				finitediff_vertex[offset + 3] = make_float4(vertex.x, vertex.y, vertex.z + finitediff_step, 1.0f);

				// д��SurfelArray
				surfel_vertex_confid[idx] = make_float4(vertex.x, vertex.y, vertex.z, depth_vertex_confid.w);
				surfel_normal_radius[idx] = make_float4(normal.x, normal.y, normal.z, depth_normal_radius.w);
				surfel_color_time[idx] = color_time;
			}
		}


		struct SurfelCandidateFilterDevice {
			//The node coordinate of the
			struct {
				const float4* live_node_coords;
				const float4* reference_node_coords;
				const DualQuaternion* node_se3;
			} warpfield_input;

			//The finite diff data input
			struct {
				DeviceArrayView<float4> vertex_finitediff_array;
				const ushort4* vertex_finitediff_knn;
				const float4* vertex_finitediff_knnweight;
				float finitediff_step;
			} vertex_input;

			//The output indicator
			mutable unsigned* candidate_validity_indicator;
			mutable ushort4* candidate_knn;
			mutable float4* candidate_knn_weight;


			__host__ __device__ __forceinline__ float min_distance2node_squared(
				const float4& vertex,
				const ushort4& knn
			) const {
				//The first knn
				float4 node = warpfield_input.live_node_coords[knn.x];
				float min_dist_square = squared_norm_xyz(node - vertex);

				//The second knn
				node = warpfield_input.live_node_coords[knn.y];
				min_dist_square = min(min_dist_square, squared_norm_xyz(node - vertex));

				//The third knn
				node = warpfield_input.live_node_coords[knn.z];
				min_dist_square = min(min_dist_square, squared_norm_xyz(node - vertex));

				//The forth knn
				node = warpfield_input.live_node_coords[knn.w];
				min_dist_square = min(min_dist_square, squared_norm_xyz(node - vertex));

				return min_dist_square;
			}

			__host__ __device__ __forceinline__ float average_distance2node_squared(
				const float4& vertex,
				const ushort4& knn
			) const {
				//The first knn
				float4 node = warpfield_input.live_node_coords[knn.x];
				float avg_dist_square = squared_norm_xyz(node - vertex);

				//The second knn
				node = warpfield_input.live_node_coords[knn.y];
				avg_dist_square += squared_norm_xyz(node - vertex);

				//The third knn
				node = warpfield_input.live_node_coords[knn.z];
				avg_dist_square += squared_norm_xyz(node - vertex);

				//The forth knn
				node = warpfield_input.live_node_coords[knn.w];
				avg_dist_square += squared_norm_xyz(node - vertex);

				//Always count for four nodes
				return 0.25f * avg_dist_square;
			}

			__host__ __device__ __forceinline__ bool is_skinning_consistent(
				const ushort4& knn
			) const {
				float live_pairwise_distance[6];
				float canonical_pairwise_distance[6];
				const unsigned short* knn_array = (const unsigned short*)&knn;
				int shift = 0;
				for (auto i = 0; i < 4; i++) {
					for (auto j = i + 1; j < 4; j++) {
						live_pairwise_distance[shift] = squared_norm_xyz(warpfield_input.live_node_coords[knn_array[i]] - warpfield_input.live_node_coords[knn_array[j]]);
						canonical_pairwise_distance[shift] = squared_norm_xyz(warpfield_input.reference_node_coords[knn_array[i]] - warpfield_input.reference_node_coords[knn_array[j]]);
						shift++;
					}
				}

				bool consistent_skinning = true;
				for (auto i = 0; i < 6; i++) {
					if (live_pairwise_distance[i] < 0.64f * canonical_pairwise_distance[i]) consistent_skinning = false;
				}
				return consistent_skinning;
			}


			__device__ __forceinline__ void processFiltering() const {
				const auto candidate_idx = threadIdx.x + blockIdx.x * blockDim.x;
				const auto offset = candidate_idx * 4;
				if (offset >= vertex_input.vertex_finitediff_array.Size()) return;

				//Load the vertex
				const float4 vertex = vertex_input.vertex_finitediff_array[offset];
				const ushort4 vertex_knn = vertex_input.vertex_finitediff_knn[offset];
				const float4 vertex_knnweight = vertex_input.vertex_finitediff_knnweight[offset];

				//The written marker
				unsigned candidate_valid = 1;

				//Check distance
				if (min_distance2node_squared(vertex, vertex_knn) >= 4.0f * NODE_RADIUS_SQUARE) candidate_valid = 0;

				//Check the consistent of skinning
				if (!is_skinning_consistent(vertex_knn)) candidate_valid = 0;

				//Check collision
				{
					//Load the data
					float4 finitediff_vertex[3], finitediff_weight[3];
					ushort4 finitediff_knn[3];
					for(auto i = 0; i < 3; i++) {
						finitediff_vertex[i] = vertex_input.vertex_finitediff_array[offset + 1 + i];
						finitediff_knn[i] = vertex_input.vertex_finitediff_knn[offset + 1 + i];
						finitediff_weight[i] = vertex_input.vertex_finitediff_knnweight[offset + 1 + i];
					}

					//Check it
					const bool compression = is_compressive_mapped(
						vertex, 
						vertex_knn, vertex_knnweight, 
						finitediff_vertex, 
						finitediff_knn, finitediff_weight, 
						warpfield_input.node_se3,
						vertex_input.finitediff_step
					);
					if (compression) candidate_valid = 0;
				}

				//Write to output
				candidate_validity_indicator[candidate_idx] = candidate_valid;
				candidate_knn[candidate_idx] = vertex_knn;
				candidate_knn_weight[candidate_idx] = vertex_knnweight;
			}
		};
	
		__global__ void filterCandidateSurfelKernel(
			const SurfelCandidateFilterDevice filter
		) {
			filter.processFiltering();
		}

	}
}


/* The method to build vertex. Using either indicator or pixels. The indicator will case sync
 */
void SparseSurfelFusion::AppendSurfelProcessor::BuildSurfelAndFiniteDiffVertex(hipStream_t stream) {
	//The size of array contains the element itself
	//����������Ԫ�ĸ���
	const size_t num_candidate = m_surfel_candidate_pixel.Size();
	m_surfel_vertex_confid.ResizeArrayOrException(num_candidate);
	m_surfel_normal_radius.ResizeArrayOrException(num_candidate);
	m_surfel_color_time.ResizeArrayOrException(num_candidate);
	m_candidate_vertex_finite_diff.ResizeArrayOrException(num_candidate * kNumFiniteDiffVertex);
	//��Ϊ֮��Ҫ�������ĸ������ݽ���skin���������ݱ��붼��0������ϵ�µ�
	

	//The appended surfel size is zero
	if (num_candidate == 0) {
		LOGGING(INFO) << "There is no appended surfel";
		return;
	}

	//Invoke the kernel
	dim3 block(64);
	dim3 grid(divUp(m_surfel_candidate_pixel.Size(), block.x));
	device::buildCandidateSurfelAndFiniteDiffVertexKernel << <grid, block, 0, stream >> > (
		m_observation,
		m_surfel_candidate_pixel,
		kFiniteDiffStep,
		//The output
		m_candidate_vertex_finite_diff.Ptr(),
		m_surfel_vertex_confid.Ptr(),
		m_surfel_normal_radius.Ptr(),
		m_surfel_color_time.Ptr()
	);
	
	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	CHECKCUDA(hipStreamSynchronize(stream));
#endif
}

void SparseSurfelFusion::AppendSurfelProcessor::SkinningFiniteDifferenceVertex(hipStream_t stream) {
	//Resize the array
	m_candidate_vertex_finitediff_knn.ResizeArrayOrException(m_candidate_vertex_finite_diff.ArraySize());
	m_candidate_vertex_finitediff_knnweight.ResizeArrayOrException(m_candidate_vertex_finite_diff.ArraySize());
	
	//If there is not surfel candidate
	if (m_candidate_vertex_finitediff_knn.ArraySize() == 0) {
		return;
	}
	//��Ҫ�õ�0������ϵ�µ�nodese3��������������ݱ��붼��0������ϵ�µ�
	m_live_node_skinner->Skinning(
		m_candidate_vertex_finite_diff.ArrayView(),
		m_candidate_vertex_finitediff_knn.ArraySlice(), m_candidate_vertex_finitediff_knnweight.ArraySlice(),
		stream
	);
	
	//Check the result of skinning: seems correct
	/*KNNSearch::CheckKNNSearch(
		m_warpfield_input.live_node_coords,
		m_candidate_vertex_finite_diff.ArrayView(),
		m_candidate_vertex_finitediff_knn.ArrayView()
	);*/
}

void SparseSurfelFusion::AppendSurfelProcessor::FilterCandidateSurfels(hipStream_t stream) {
	//Resize the indicator
	m_candidate_surfel_validity_indicator.ResizeArrayOrException(m_surfel_candidate_pixel.Size());
	m_surfel_knn.ResizeArrayOrException(m_surfel_candidate_pixel.Size());
	m_surfel_knn_weight.ResizeArrayOrException(m_surfel_candidate_pixel.Size());
	
	//Check if the size is zero
	if(m_surfel_knn.ArraySize() == 0) return;
	
	//Construct the filter
	device::SurfelCandidateFilterDevice filter;
	
	filter.warpfield_input.live_node_coords = m_warpfield_input.live_node_coords.RawPtr();
	filter.warpfield_input.reference_node_coords = m_warpfield_input.reference_node_coords.RawPtr();
	filter.warpfield_input.node_se3 = m_warpfield_input.node_se3.RawPtr();
	
	filter.vertex_input.vertex_finitediff_array = m_candidate_vertex_finite_diff.ArrayView();
	filter.vertex_input.vertex_finitediff_knn = m_candidate_vertex_finitediff_knn.Ptr();
	filter.vertex_input.vertex_finitediff_knnweight = m_candidate_vertex_finitediff_knnweight.Ptr();
	filter.vertex_input.finitediff_step = kFiniteDiffStep;
	
	filter.candidate_validity_indicator = m_candidate_surfel_validity_indicator.Ptr();
	filter.candidate_knn = m_surfel_knn.Ptr();
	filter.candidate_knn_weight = m_surfel_knn_weight.Ptr();
	
	//Seems now ready for device code
	dim3 blk(64);
	dim3 grid(divUp(m_surfel_candidate_pixel.Size(), blk.x));
	device::filterCandidateSurfelKernel << <grid, blk, 0, stream >> > (filter);
	
	//Do a prefix sum on the indicator
	m_candidate_surfel_validity_prefixsum.InclusiveSum(m_candidate_surfel_validity_indicator.ArrayView(), stream);

}


