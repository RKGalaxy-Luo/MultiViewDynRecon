#include "hip/hip_runtime.h"
/*****************************************************************//**
 * \file   ComputeTriangleIndices.cu
 * \brief  �����޸��������񣬹�������
 * 
 * \author LUOJIAXUAN
 * \date   June 3rd 2024
 *********************************************************************/
#include "ComputeTriangleIndices.h"
#if defined(__HIPCC__)		//�����NVCC����������
#include <hipcub/hipcub.hpp>
#endif
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include "DeviceConstants.cuh"
namespace SparseSurfelFusion {
    /**
     * \brief �����Ƿ�ϸ�ֽڵ㣬ϸ�ֵ��������ýڵ�û�к���(��û�б�ϸ�ֹ�)�����ҽڵ�cube�д��������λ�������ཻ����������ڵ�ϸ��.
     */
    struct ifSubdivide {
        __device__ bool operator()(const OctNode& x) {
            return (x.children[0] == -1 && x.children[1] == -1 && x.children[2] == -1 && x.children[3] == -1 && x.children[4] == -1 && x.children[5] == -1 && x.children[6] == -1 && x.children[7] == -1) && (x.hasTriangle || x.hasIntersection);
            //return (x.children[0] == -1) && (x.hasTriangle || x.hasIntersection);
        }
    };
}

__global__ void SparseSurfelFusion::device::ComputeVertexImplicitFunctionValueKernel(DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> BaseFunctions, DeviceArrayView<float> dx, DeviceArrayView<int> encodeNodeIndexInFunction, const unsigned int VertexArraySize, const float isoValue, float* vvalue)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= VertexArraySize)	return;
    VertexNode nowVertex = VertexArray[idx];
    int depth = nowVertex.depth;
    float val = 0.0f;
    int exceedChildrenId = device::childrenVertexKind[nowVertex.vertexKind];
    int nowNode = nowVertex.ownerNodeIdx;
    if (nowNode > 0) {
        // ����vertex��owner node���ھ� -> owner node���ڵ㼰���ڵ��ھ� -> ֱ����ǰ�ڵ�Ϊ��
        while (nowNode != -1) {
            for (int i = 0; i < 27; i++) {
                int neighbor = NodeArray[nowNode].neighs[i];
                if (neighbor != -1) {
                    int idxO[3];
                    int encode_idx = encodeNodeIndexInFunction[neighbor];
                    idxO[0] = encode_idx % decodeOffset_1;
                    idxO[1] = (encode_idx / decodeOffset_1) % decodeOffset_1;
                    idxO[2] = encode_idx / decodeOffset_2;

                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcX = BaseFunctions[idxO[0]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcY = BaseFunctions[idxO[1]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcZ = BaseFunctions[idxO[2]];

                    val += dx[neighbor] * value(funcX, nowVertex.pos.coords[0]) * value(funcY, nowVertex.pos.coords[1]) * value(funcZ, nowVertex.pos.coords[2]);
                }
            }
            nowNode = NodeArray[nowNode].parent;
        }
        nowNode = nowVertex.ownerNodeIdx;
        while (depth < device::maxDepth) {
            depth++;
            nowNode = NodeArray[nowNode].children[exceedChildrenId];
            if (nowNode == -1) break;
            for (int i = 0; i < 27; i++) {
                int neighbor = NodeArray[nowNode].neighs[i];
                if (neighbor != -1) {
                    int idxO[3];
                    int encode_idx = encodeNodeIndexInFunction[neighbor];
                    idxO[0] = encode_idx % decodeOffset_1;
                    idxO[1] = (encode_idx / decodeOffset_1) % decodeOffset_1;
                    idxO[2] = encode_idx / decodeOffset_2;

                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcX = BaseFunctions[idxO[0]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcY = BaseFunctions[idxO[1]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcZ = BaseFunctions[idxO[2]];

                    val += dx[neighbor] * value(funcX, nowVertex.pos.coords[0]) * value(funcY, nowVertex.pos.coords[1]) * value(funcZ, nowVertex.pos.coords[2]);
                }
            }
        }
    }
    vvalue[idx] = val - isoValue;
}

__global__ void SparseSurfelFusion::device::generateVertexNumsKernel(DeviceArrayView<EdgeNode> EdgeArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> vvalue, const unsigned int EdgeArraySize, int* vexNums, bool* markValidVertex)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= EdgeArraySize)	return;
    EdgeNode nowEdge = EdgeArray[idx];
    int owner = nowEdge.ownerNodeIdx;
    int kind = nowEdge.edgeKind;
    int index[2];
    index[0] = device::edgeVertex[kind][0];// ���ݱߵ�λ��˳���ҵ���Ӧ�ı�
    index[1] = device::edgeVertex[kind][1];// ���ݱߵ�λ��˳���ҵ���Ӧ�ı�

    int v1 = NodeArray[owner].vertices[index[0]] - 1; // �ҵ��߶�Ӧ��������
    int v2 = NodeArray[owner].vertices[index[1]] - 1; // �ҵ��߶�Ӧ��������
    if (vvalue[v1] * vvalue[v2] <= 0) { // ��������������������ֵ��ţ�˵�����洩����������֮��
        vexNums[idx] = 1;               // ��¼�����
        markValidVertex[idx] = true;
    }
    else {
        vexNums[idx] = 0;
        markValidVertex[idx] = false;
    }
}

__global__ void SparseSurfelFusion::device::generateTriangleNumsKernel(DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> vvalue, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, int* triNums, int* cubeCatagory)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;
    OctNode currentNode = NodeArray[offset];    // ��ǰ����ڵ�
    int currentCubeCatagory = 0;                // ����������
    for (int i = 0; i < 8; i++) {
        if (vvalue[currentNode.vertices[i] - 1] < 0) {  // ��ǰ�ڵ�Ķ�����������
            currentCubeCatagory |= 1 << i;  // ͨ���ж���Щ���������ڲ����Ӷ��ж�������256���еĵڼ�������
        }
    }
    triNums[idx] = device::trianglesCount[currentCubeCatagory]; // ���ֱ�Ӳ鵽����
    cubeCatagory[idx] = currentCubeCatagory;    // ��¼��ǰ�ڵ������������
}

__global__ void SparseSurfelFusion::device::generateIntersectionPoint(DeviceArrayView<OctNode> NodeArray, DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<float> vvalue, const EdgeNode* validEdgeArray, const int* validVexAddress, const unsigned int validEdgeArraySize, Point3D<float>* VertexBuffer)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= validEdgeArraySize)	return;
    int owner = validEdgeArray[idx].ownerNodeIdx;   // ��ǰ��������һ���ڵ�
    int kind = validEdgeArray[idx].edgeKind;        // ��ǰ������һ�����͵ı�(��cube�е�λ��)
    int orientation = kind >> 2;    // ������x����y����z

    int index[2];

    index[0] = edgeVertex[kind][0];
    index[1] = edgeVertex[kind][1];

    int v1 = NodeArray[owner].vertices[index[0]] - 1;
    int v2 = NodeArray[owner].vertices[index[1]] - 1;
    Point3D<float> p1 = VertexArray[v1].pos;  // �ߵĶ���1
    Point3D<float> p2 = VertexArray[v2].pos;  // �ߵĶ���2
    float f1 = vvalue[v1];      // ����1��������
    float f2 = vvalue[v2];      // ����2��������
    Point3D<float> isoPoint;    // �����ؽ�����Ľ�������
    interpolatePoint(p1, p2, orientation, f1, f2, isoPoint);
    VertexBuffer[validVexAddress[idx]] = isoPoint;  // ������ؽ������ཻ�Ľ���
}

__device__ void SparseSurfelFusion::device::interpolatePoint(const Point3D<float>& p1, const Point3D<float>& p2, const int& dim, const float& v1, const float& v2, Point3D<float>& out)
{
    for (int i = 0; i < 3; i++) {
        if (i != dim) {// ������Ҫ�����ά��(x, y, z)��˵����ǰ������ֵ���Բ��ñ�
            out.coords[i] = p1.coords[i];
        }
    }
    float pivot = v1 / (v1 - v2);   // ��������������ߵ��ĸ����㣬�ؽ�����ĵ�����ľ�������������0��ľ���ɱ���
    float anotherPivot = 1 - pivot;
    out.coords[dim] = p2.coords[dim] * pivot + p1.coords[dim] * anotherPivot;
}

__global__ void SparseSurfelFusion::device::generateTrianglePos(DeviceArrayView<OctNode> NodeArray, DeviceArrayView<FaceNode> FaceArray, DeviceArrayView<int> triNums, DeviceArrayView<int> cubeCatagory, DeviceArrayView<int> vexAddress, DeviceArrayView<int> triAddress, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, TriangleIndex* TriangleBuffer, int* hasSurfaceIntersection)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;                             // ����maxDepth��ڵ�
    OctNode currentNode = NodeArray[offset];                                    // ��ǰ�ڵ�
    int currentTriNum = triNums[idx];                                           // ��ǰ����ڵ�cube��Ӧ������������
    int currentCubeCatagory = cubeCatagory[idx];                                // ��ǰ����ڵ��cube����
    int currentTriangleBufferStart = triAddress[idx];                       // ��ǰ�ڵ��Ӧ�����ε�indexƫ�ƣ�ÿ����������Ҫ��3�������index
    bool edgeHasVertex[12] = {  false, false, false, false, false, false,
                                false, false, false, false, false, false };     // ���Ƿ��ж���

    for (int i = 0; i < currentTriNum; i++) {                            // ������ǰ�ڵ�cube��Ӧ������������
        int edgeIdx[3];                     // ���������ϵĵ㹹�ɵ�������
        edgeIdx[0] = device::triangles[currentCubeCatagory][3 * i];
        edgeIdx[1] = device::triangles[currentCubeCatagory][3 * i + 1];
        edgeIdx[2] = device::triangles[currentCubeCatagory][3 * i + 2];

        edgeHasVertex[edgeIdx[0]] = true;   // ����Щ�߱��Ϊ�ж���ı�
        edgeHasVertex[edgeIdx[1]] = true;   // ����Щ�߱��Ϊ�ж���ı�
        edgeHasVertex[edgeIdx[2]] = true;   // ����Щ�߱��Ϊ�ж���ı�

        int vertexIdx[3];
        vertexIdx[0] = vexAddress[currentNode.edges[edgeIdx[0]] - 1];       // ȷ��������������εĶ��������
        vertexIdx[1] = vexAddress[currentNode.edges[edgeIdx[1]] - 1];       // ȷ��������������εĶ��������
        vertexIdx[2] = vexAddress[currentNode.edges[edgeIdx[2]] - 1];       // ȷ��������������εĶ��������

        TriangleBuffer[currentTriangleBufferStart + i].idx[0] = vertexIdx[0];   // ��������������TriangleBuffer
        TriangleBuffer[currentTriangleBufferStart + i].idx[1] = vertexIdx[1];   // ��������������TriangleBuffer
        TriangleBuffer[currentTriangleBufferStart + i].idx[2] = vertexIdx[2];   // ��������������TriangleBuffer
    }
    int currentFace;
    int parentNodeIndex;
    for (int i = 0; i < 6; i++) {       // �����ڵ�cube��6����
        bool mark = false;              // ��¼�Ƿ����Surface-Edge Intersections(����ཻ)
        for (int j = 0; j < 4; j++) {   // ����ĳ�����4����
            mark |= edgeHasVertex[device::faceEdges[i][j]]; // ֻҪ����һ��true����mark = true����������ཻ
        }
        if (mark == true) {             // ��������ཻ
            parentNodeIndex = NodeArray[offset].parent;                 // ��ǰ�ڵ�ĸ��ڵ�
            currentFace = currentNode.faces[i] - 1;                     // ��ǰ�ڵ�����"����ཻ"����
            hasSurfaceIntersection[currentFace] = 1;                    // ����ǰ���Ϊ����ཻ
            while (FaceArray[currentFace].hasParentFace != -1) {        // һֱ���ϲ�ڵ�������鿴�Ƿ����"Parent Face"
                currentFace = NodeArray[parentNodeIndex].faces[i] - 1;  // ���׽ڵ��������͵��� ���뺢�ӽڵ������ͬһ��index
                parentNodeIndex = NodeArray[parentNodeIndex].parent;    // �ٻ�õ�ǰ�ڵ�ĸ��׽ڵ�
                hasSurfaceIntersection[currentFace] = 1;                // �����ڵ�������Ҳ����Ϊ"����ཻ"
            }
        }
    }
}

__global__ void SparseSurfelFusion::device::generateSubdivideTrianglePos(const EasyOctNode* SubdivideArray, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, const int* SubdivideTriNums, const int* SubdivideCubeCatagory, const int* SubdivideVexAddress, const int* SubdivideTriAddress, TriangleIndex* SubdivideTriangleBuffer)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;
    int nowTriNum = SubdivideTriNums[idx];
    int nowCubeCatagory = SubdivideCubeCatagory[idx];
    int nowTriangleBufferStart = SubdivideTriAddress[idx];

    for (int i = 0; i < nowTriNum; i++) {
        int edgeIdx[3];
        edgeIdx[0] = triangles[nowCubeCatagory][3 * i];
        edgeIdx[1] = triangles[nowCubeCatagory][3 * i + 1];
        edgeIdx[2] = triangles[nowCubeCatagory][3 * i + 2];

        int vertexIdx[3];
        vertexIdx[0] = SubdivideVexAddress[SubdivideArray[offset].edges[edgeIdx[0]] - 1];
        vertexIdx[1] = SubdivideVexAddress[SubdivideArray[offset].edges[edgeIdx[1]] - 1];
        vertexIdx[2] = SubdivideVexAddress[SubdivideArray[offset].edges[edgeIdx[2]] - 1];

        SubdivideTriangleBuffer[nowTriangleBufferStart + i].idx[0] = vertexIdx[0];
        SubdivideTriangleBuffer[nowTriangleBufferStart + i].idx[1] = vertexIdx[1];
        SubdivideTriangleBuffer[nowTriangleBufferStart + i].idx[2] = vertexIdx[2];
    }
}

__global__ void SparseSurfelFusion::device::ProcessLeafNodesAtOtherDepth(DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<float> vvalue, const unsigned int OtherDepthNodeCount, const int* hasSurfaceIntersection, OctNode* NodeArray, bool* markValidSubdividedNode)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= OtherDepthNodeCount)	return;
    OctNode currentNode = NodeArray[idx];
    int hasTri = 0;
    int sign = (vvalue[currentNode.vertices[0] - 1] < 0) ? -1 : 1;
    for (int i = 1; i < 8; i++) {
        if (sign * vvalue[currentNode.vertices[i] - 1] < 0) {
            hasTri = 1;
            break;
        }
    }
    NodeArray[idx].hasTriangle = hasTri;

    int hasIntersection = 0;
    for (int i = 0; i < 6; i++) {
        if (hasSurfaceIntersection[currentNode.faces[i] - 1]) {
            hasIntersection = 1;
            break;
        }
    }
    NodeArray[idx].hasIntersection = hasIntersection;

    if ((NodeArray[idx].children[0] == -1) && (hasTri || hasIntersection)) {
        markValidSubdividedNode[idx] = true;
    }
    else {
        markValidSubdividedNode[idx] = false;
    }
}

__global__ void SparseSurfelFusion::device::precomputeSubdivideDepth(DeviceArrayView<OctNode> SubdivideNode, DeviceArrayView<unsigned int> DepthBuffer, const int SubdivideNum, int* SubdivideDepthBuffer, int* SubdivideDepthNum)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideNum)	return;
    int nodeIndex = SubdivideNode[idx].neighs[13];
    int depth = DepthBuffer[nodeIndex];
    SubdivideDepthBuffer[idx] = depth;
    SubdivideDepthNum[idx + depth * SubdivideNum] = 1;
}

__global__ void SparseSurfelFusion::device::singleRebuildArray(DeviceArrayView<OctNode> SubdivideNode, DeviceArrayView<int> SubdivideDepthBuffer, const unsigned int iterRound, const unsigned int NodeArraySize, const unsigned int SubdivideArraySize, EasyOctNode* SubdivideArray, int* SubdivideArrayDepthBuffer, Point3D<float>* SubdivideArrayCenterBuffer)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideArraySize)	return;
    int rootId = SubdivideNode[iterRound].neighs[13];       // ��ǰ�ڵ���NodeArray�е�λ��
    int rootDepth = SubdivideDepthBuffer[iterRound];        // ��ǰ�ڵ�����
    int rootKey = SubdivideNode[iterRound].key;             // ��ǰ�ڵ��key
    int thisNodeDepth = getSubdivideDepth(rootDepth, idx);  // ��ǰ�ڵ���Octree�еľ������
    int relativeDepth = thisNodeDepth - rootDepth;          // ��ǰ�ڵ���Octree�������ϸ��root��������
    int idxOffset = idx - (powf(8, relativeDepth) - 1) / 7; // �ڵ��ڵ�ǰ(���)���еĵڼ�������ǰ���һ��idxOffset = 0

    if (thisNodeDepth < maxDepth) {                         // �������һ��
        int nextDepthAddress = (powf(8, relativeDepth + 1) - 1) / 7;    // ��һ��(���)���ƫ��
        for (int k = 0; k < 8; ++k) {                       // �������ǰϸ�ֽڵ㺢�ӵ����������ǽ���NodeArraySize�����
            SubdivideArray[idx].children[k] = NodeArraySize + nextDepthAddress + (idxOffset << 3) + k;
        }
    }
    else {
        for (int k = 0; k < 8; ++k) {                       // ���һ��
            SubdivideArray[idx].children[k] = -1;           // �����Ǻ��ӽڵ�
        }
    }

    if (idx != 0) { // ���������Ե�root�ڵ�(��Ե�0��)������ڵ�ĸ��ڵ�index
        int parentDepthAddress = (powf(8, relativeDepth - 1) - 1) / 7;
        SubdivideArray[idx].parent = NodeArraySize + parentDepthAddress + (idxOffset >> 3);
    }

    int thisKey = rootKey;
    thisKey |= (idxOffset) << (3 * (maxDepth - thisNodeDepth)); // ���쵱ǰ�������ڵ��key��root��key��ǰ׺��
    SubdivideArray[idx].key = thisKey;

    SubdivideArrayDepthBuffer[idx] = thisNodeDepth;             // ��ǰ�ڵ�ľ������
    Point3D<float> thisNodeCenter;
    getNodeCenterAllDepth(thisKey, thisNodeDepth, thisNodeCenter);
    SubdivideArrayCenterBuffer[idx] = thisNodeCenter;           // ��ǰ�ڵ����ά����
}

__device__ int SparseSurfelFusion::device::getSubdivideDepth(const int& rootDepth, const int& idx)
{
    int up = idx * 7 + 1;
    int base = 8;
    int relativeDepth = 0;
    while (base <= up) {
        relativeDepth++;
        base <<= 3;
    }
    return rootDepth + relativeDepth;
}

__device__ void SparseSurfelFusion::device::getNodeCenterAllDepth(const int& key, const int& currentDepth, Point3D<float>& center)
{
    center.coords[0] = float(0.5);
    center.coords[1] = float(0.5);
    center.coords[2] = float(0.5);
    float Width = 0.25f;
    for (int i = device::maxDepth - 1; i >= (device::maxDepth - currentDepth); --i) {
        if ((key >> (3 * i + 2)) & 1) center.coords[0] += Width;
        else center.coords[0] -= Width;

        if ((key >> (3 * i + 1)) & 1) center.coords[1] += Width;
        else center.coords[1] -= Width;

        if ((key >> (3 * i)) & 1) center.coords[2] += Width;
        else center.coords[2] -= Width;

        Width /= 2;
    }
}

__global__ void SparseSurfelFusion::device::computeRebuildNeighbor(DeviceArrayView<OctNode> NodeArray, const unsigned int currentLevelOffset, const unsigned int currentLevelNodesCount, const unsigned int NodeArraySize, const unsigned int depth, EasyOctNode* SubdivideArray)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= currentLevelNodesCount)	return;
    const unsigned int offset = currentLevelOffset + idx;   // ��ǰ��ڵ���SubdivideArray�е�ƫ��
    for (int i = 0; i < 27; i++) {
        int sonKey = (SubdivideArray[offset].key >> (3 * (device::maxDepth - depth))) & 7;  // ��ǰ�ڵ���ǵ�ǰ��ĵڼ��Žڵ�
        int parentIdx = SubdivideArray[offset].parent;      // ���ϸ�ֽڵ�ĸ��ڵ�
        int neighParent;                                    // ������(0-7��)�ڵ�ĵ�i���ھӣ�������ڵ�ĸ��ڵ�ĵڼ����ھ�֮�У������index
        if (parentIdx < NodeArraySize) {                    // ϸ�ֽڵ㸸�ڵ�ΪNodeArray�е�ֵ
            neighParent = NodeArray[parentIdx].neighs[device::LUTparent[sonKey][i]];
        }
        else {                                              // ϸ�ֽڵ㸸�ڵ��ǲŲ����ֵ
            neighParent = SubdivideArray[parentIdx - NodeArraySize].neighs[device::LUTparent[sonKey][i]];
        }
        if (neighParent != -1) {    // ϸ�ֽڵ��i���ھ��ǿ������丸�ڵ��ھ��ж�λ����
            if (neighParent < NodeArraySize) {
                SubdivideArray[offset].neighs[i] = NodeArray[neighParent].children[device::LUTchild[sonKey][i]];
            }
            else {
                SubdivideArray[offset].neighs[i] = SubdivideArray[neighParent - NodeArraySize].children[device::LUTchild[sonKey][i]];
            }
        }
        else {                      // ϸ�ֽڵ㸸�ڵ�Ϊ��
            SubdivideArray[offset].neighs[i] = -1;
        }
    }
}

__global__ void SparseSurfelFusion::device::initSubdivideVertexOwner(const EasyOctNode* SubdivideArray, const Point3D<float>* SubdivideArrayCenterBuffer, const unsigned int currentLevelOffset, const unsigned int currentLevelNodesCount, const unsigned int NodeArraySize, VertexNode* SubdividePreVertexArray, bool* markValidSubdivideVertex)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= currentLevelNodesCount)	return;
    const unsigned int offset = currentLevelOffset + idx;
    int NodeOwnerKey[8] = { device::maxIntValue,device::maxIntValue, device::maxIntValue, device::maxIntValue,
                            device::maxIntValue, device::maxIntValue, device::maxIntValue, device::maxIntValue };
    int NodeOwnerIdx[8] = { -1, -1, -1, -1, -1, -1, -1, -1 };
    int depth = maxDepth;
    float halfWidth = 1.0f / (1 << (depth + 1));
    float Width = 1.0f / (1 << depth);
    float Widthsq = Width * Width;
    Point3D<float> neighCenter[27];
    int neigh[27];
#pragma unroll
    for (int i = 0; i < 27; i++) {
        neigh[i] = SubdivideArray[offset].neighs[i];
        if (neigh[i] != -1 && neigh[i] >= NodeArraySize) {
            neighCenter[i] = SubdivideArrayCenterBuffer[neigh[i] - NodeArraySize];
        }
    }
    const Point3D<float>& nodeCenter = neighCenter[13];

    Point3D<float> vertexPos[8];
#pragma unroll
    for (int i = 0; i < 8; i++) {
        vertexPos[i].coords[0] = nodeCenter.coords[0] + (2 * (i & 1) - 1) * halfWidth;
        vertexPos[i].coords[1] = nodeCenter.coords[1] + (2 * ((i & 2) >> 1) - 1) * halfWidth;
        vertexPos[i].coords[2] = nodeCenter.coords[2] + (2 * ((i & 4) >> 2) - 1) * halfWidth;
    }

#pragma unroll

    for (int i = 0; i < 8; i++) {       // ����8��vertex
        for (int j = 0; j < 27; j++) {  // ����27���ھӽڵ�
            if (neigh[j] != -1 && SquareDistance(vertexPos[i], neighCenter[j]) < Widthsq) { // �����vertex����Ч�ڵ�
                int neighKey;
                if (neigh[j] < NodeArraySize) continue; // ����ڵ��ھ��Ѿ����������������Σ�������
                else
                    neighKey = SubdivideArray[neigh[j] - NodeArraySize].key;
                if (NodeOwnerKey[i] > neighKey) {
                    NodeOwnerKey[i] = neighKey;
                    NodeOwnerIdx[i] = neigh[j];
                }
            }
        }
    }
#pragma unroll
    for (int i = 0; i < 8; i++) {
        int vertexIdx = 8 * idx + i;
        if (NodeOwnerIdx[i] == NodeArraySize + offset) {    // �����owner node���Լ�
            SubdividePreVertexArray[vertexIdx].ownerNodeIdx = NodeOwnerIdx[i];
            SubdividePreVertexArray[vertexIdx].pos.coords[0] = vertexPos[i].coords[0];
            SubdividePreVertexArray[vertexIdx].pos.coords[1] = vertexPos[i].coords[1];
            SubdividePreVertexArray[vertexIdx].pos.coords[2] = vertexPos[i].coords[2];
            SubdividePreVertexArray[vertexIdx].vertexKind = i;
            SubdividePreVertexArray[vertexIdx].depth = depth;
            markValidSubdivideVertex[vertexIdx] = true;
        }
        else {
            markValidSubdivideVertex[vertexIdx] = false;
        }
    }

}

__global__ void SparseSurfelFusion::device::maintainSubdivideVertexNodePointer(DeviceArrayView<Point3D<float>> CenterBuffer, const unsigned int VertexArraySize, const unsigned int NodeArraySize, const Point3D<float>* SubdivideArrayCenterBuffer, VertexNode* VertexArray, EasyOctNode* SubdivideArray)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= VertexArraySize)	return;
    int owner = VertexArray[idx].ownerNodeIdx;
    float Width = 1.0f / (1 << device::maxDepth);
    float WidthSquare = Width * Width;
    Point3D<float> neighCenter[27];
    Point3D<float> vertexPos = VertexArray[idx].pos;

    int neigh[27];
    for (int i = 0; i < 27; i++) {
        neigh[i] = SubdivideArray[owner - NodeArraySize].neighs[i];
    }
    for (int i = 0; i < 27; i++) {
        if (neigh[i] != -1) {
            if (neigh[i] < NodeArraySize) {
                neighCenter[i] = CenterBuffer[neigh[i]];
            }
            else {
                neighCenter[i] = SubdivideArrayCenterBuffer[neigh[i] - NodeArraySize];
            }
        }
    }
    int count = 0;
    for (int i = 0; i < 27; i++) {
        if (neigh[i] != -1 && SquareDistance(vertexPos, neighCenter[i]) < WidthSquare) {
            VertexArray[idx].nodes[count] = neigh[i];
            count++;
            int index = 0;
            if (neighCenter[i].coords[0] - vertexPos.coords[0] < 0) index |= 1;
            if (neighCenter[i].coords[2] - vertexPos.coords[2] < 0) index |= 4;
            if (neighCenter[i].coords[1] - vertexPos.coords[1] < 0) {
                if (index & 1) {
                    index += 1;
                }
                else {
                    index += 3;
                }
            }
            if (neigh[i] >= NodeArraySize) {
                SubdivideArray[neigh[i] - NodeArraySize].vertices[index] = idx + 1;
            }
        }
    }
}

__global__ void SparseSurfelFusion::device::initSubdivideEdgeArray(const EasyOctNode* SubdivideArray, const Point3D<float>* SubdivideArrayCenterBuffer, const unsigned int NodeArraySize, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, EdgeNode* SubdividePreEdgeArray, bool* markValidSubdivideEdge)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;
    int NodeOwnerKey[12] = { device::maxIntValue, device::maxIntValue, device::maxIntValue,
                             device::maxIntValue, device::maxIntValue, device::maxIntValue,
                             device::maxIntValue, device::maxIntValue, device::maxIntValue,
                             device::maxIntValue, device::maxIntValue, device::maxIntValue };
    int NodeOwnerIdx[12] = { -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 };
    float halfWidth = 1.0f / (1 << (device::maxDepth + 1));
    float Width = 1.0f / (1 << device::maxDepth);
    float WidthSquare = Width * Width;
    Point3D<float> neighCenter[27];
    int neigh[27];
#pragma unroll
    for (int i = 0; i < 27; i++) {
        neigh[i] = SubdivideArray[offset].neighs[i];
        if (neigh[i] != -1 && neigh[i] >= NodeArraySize) {
            neighCenter[i] = SubdivideArrayCenterBuffer[neigh[i] - NodeArraySize];
        }
    }
    const Point3D<float>& nodeCenter = neighCenter[13];
    Point3D<float> edgeCenterPos[12];
    int orientation[12];
    int off[24];
#pragma unroll
    for (int i = 0; i < 12; i++) {
        orientation[i] = i >> 2;
        off[2 * i] = i & 1;
        off[2 * i + 1] = (i & 2) >> 1;
        int multi[3];
        int dim = 2 * i;
        for (int j = 0; j < 3; j++) {
            if (orientation[i] == j) {
                multi[j] = 0;
            }
            else {
                multi[j] = (2 * off[dim] - 1);
                dim++;
            }
        }
        edgeCenterPos[i].coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
        edgeCenterPos[i].coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
        edgeCenterPos[i].coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;
    }

#pragma unroll
    for (int i = 0; i < 12; i++) {
        for (int j = 0; j < 27; j++) {
            if (neigh[j] != -1 && SquareDistance(edgeCenterPos[i], neighCenter[j]) < WidthSquare) {
                int neighKey;
                if (neigh[j] < NodeArraySize) continue;
                else
                    neighKey = SubdivideArray[neigh[j] - NodeArraySize].key;
                if (NodeOwnerKey[i] > neighKey) {
                    NodeOwnerKey[i] = neighKey;
                    NodeOwnerIdx[i] = neigh[j];
                }
            }
        }
    }
#pragma unroll
    for (int i = 0; i < 12; i++) {
        int edgeIdx = 12 * idx + i;
        if (NodeOwnerIdx[i] == offset + NodeArraySize) {
            SubdividePreEdgeArray[edgeIdx].ownerNodeIdx = NodeOwnerIdx[i];
            SubdividePreEdgeArray[edgeIdx].edgeKind = i;
            markValidSubdivideEdge[edgeIdx] = true;
        }
        else {
            markValidSubdivideEdge[edgeIdx] = false;
        }
    }
}

__global__ void SparseSurfelFusion::device::maintainSubdivideEdgeNodePointer(DeviceArrayView<Point3D<float>> CenterBuffer, const Point3D<float>* SubdivideArrayCenterBuffer, const unsigned int EdgeArraySize, const unsigned int NodeArraySize, EasyOctNode* SubdivideArray, EdgeNode* EdgeArray)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= EdgeArraySize)	return;
    int owner = EdgeArray[idx].ownerNodeIdx;

    int depth = device::maxDepth;
    float halfWidth = 1.0f / (1 << (depth + 1));
    float Width = 1.0f / (1 << depth);
    float WidthSquare = Width * Width;

    Point3D<float> neighCenter[27];
    int neighbor[27];
    for (int i = 0; i < 27; i++) {
        neighbor[i] = SubdivideArray[owner - NodeArraySize].neighs[i];
        if (neighbor[i] != -1) {
            if (neighbor[i] < NodeArraySize)
                neighCenter[i] = CenterBuffer[neighbor[i]];
            else
                neighCenter[i] = SubdivideArrayCenterBuffer[neighbor[i] - NodeArraySize];
        }
    }

    const Point3D<float>& nodeCenter = neighCenter[13];
    Point3D<float> edgeCenterPos;
    int multi[3];
    int dim = 0;
    int orientation = EdgeArray[idx].edgeKind >> 2;
    int off[2];
    off[0] = EdgeArray[idx].edgeKind & 1;
    off[1] = (EdgeArray[idx].edgeKind & 2) >> 1;
    for (int i = 0; i < 3; i++) {
        if (orientation == i) {
            multi[i] = 0;
        }
        else {
            multi[i] = (2 * off[dim] - 1);
            dim++;
        }
    }
    edgeCenterPos.coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
    edgeCenterPos.coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
    edgeCenterPos.coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;

    int count = 0;
    for (int i = 0; i < 27; i++) {
        if (neighbor[i] != -1 && SquareDistance(edgeCenterPos, neighCenter[i]) < WidthSquare) {
            EdgeArray[idx].nodes[count] = neighbor[i];
            count++;
            int index = orientation << 2;
            int dim = 0;
            for (int j = 0; j < 3; j++) {
                if (orientation != j) {
                    if (neighCenter[i].coords[j] - edgeCenterPos.coords[j] < 0) index |= (1 << dim);
                    dim++;
                }
            }
            if (neighbor[i] >= NodeArraySize)
                SubdivideArray[neighbor[i] - NodeArraySize].edges[index] = idx + 1;
        }
    }
}

__forceinline__ __device__ double SparseSurfelFusion::device::SquareDistance(const Point3D<float>& p1, const Point3D<float>& p2)
{
    return (p1.coords[0] - p2.coords[0]) * (p1.coords[0] - p2.coords[0]) + (p1.coords[1] - p2.coords[1]) * (p1.coords[1] - p2.coords[1]) + (p1.coords[2] - p2.coords[2]) * (p1.coords[2] - p2.coords[2]);
}

__global__ void SparseSurfelFusion::device::computeSubdivideVertexImplicitFunctionValue(const VertexNode* SubdivideVertexArray, const EasyOctNode* SubdivideArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> dx, DeviceArrayView<int> EncodedNodeIdxInFunction, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> baseFunctions, const unsigned int NodeArraySize, const unsigned int rootId, const unsigned int SubdivideVertexArraySize, const float isoValue, float* SubdivideVvalue)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideVertexArraySize)	return;
    VertexNode nowVertex = SubdivideVertexArray[idx];
    int depth = device::maxDepth;
    float val = 0.0f;
    int nowNode = nowVertex.ownerNodeIdx;
    if (nowNode > 0) {
        while (nowNode != -1) {
            for (int k = 0; k < 27; ++k) {
                int neigh;
                if (nowNode < NodeArraySize)
                    neigh = NodeArray[nowNode].neighs[k];
                else
                    neigh = SubdivideArray[nowNode - NodeArraySize].neighs[k];
                if (neigh != -1) {
                    if (neigh == NodeArraySize)
                        neigh = rootId;
                    int idxO[3];
                    int encode_idx;
                    if (neigh < NodeArraySize)
                        encode_idx = EncodedNodeIdxInFunction[neigh];
                    else continue;  // d_x = 0 in Subdivide space
                    idxO[0] = encode_idx % decodeOffset_1;
                    idxO[1] = (encode_idx / decodeOffset_1) % decodeOffset_1;
                    idxO[2] = encode_idx / decodeOffset_2;

                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcX = baseFunctions[idxO[0]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcY = baseFunctions[idxO[1]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcZ = baseFunctions[idxO[2]];

                    val += dx[neigh] * value(funcX, nowVertex.pos.coords[0]) * value(funcY, nowVertex.pos.coords[1]) * value(funcZ, nowVertex.pos.coords[2]);
                }
            }
            if (nowNode < NodeArraySize)
                nowNode = NodeArray[nowNode].parent;
            else
                nowNode = SubdivideArray[nowNode - NodeArraySize].parent;
        }
    }
    SubdivideVvalue[idx] = val - isoValue;
}

__global__ void SparseSurfelFusion::device::computeSubdivideVertexImplicitFunctionValue(const VertexNode* SubdivideVertexArray, const EasyOctNode* SubdivideArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> dx, DeviceArrayView<int> EncodedNodeIdxInFunction, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> baseFunctions, const unsigned int NodeArraySize, const int* ReplacedNodeId, const int* IsRoot, const unsigned int SubdivideVertexArraySize, const float isoValue, float* SubdivideVvalue)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideVertexArraySize)	return;
    VertexNode nowVertex = SubdivideVertexArray[idx];
    float val = 0.0f;
    int nowNode = nowVertex.ownerNodeIdx;
    if (nowNode > 0) {
        while (nowNode != -1) {
            for (int k = 0; k < 27; ++k) {
                int neigh;
                if (nowNode < NodeArraySize)
                    neigh = NodeArray[nowNode].neighs[k];
                else
                    neigh = SubdivideArray[nowNode - NodeArraySize].neighs[k];
                if (neigh != -1) {
                    if (neigh >= NodeArraySize && IsRoot[neigh - NodeArraySize])
                        neigh = ReplacedNodeId[neigh - NodeArraySize];
                    int idxO[3];
                    int encode_idx;
                    if (neigh < NodeArraySize)
                        encode_idx = EncodedNodeIdxInFunction[neigh];
                    else continue;  // d_x = 0 in Subdivide space
                    idxO[0] = encode_idx % decodeOffset_1;
                    idxO[1] = (encode_idx / decodeOffset_1) % decodeOffset_1;
                    idxO[2] = encode_idx / decodeOffset_2;

                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcX = baseFunctions[idxO[0]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcY = baseFunctions[idxO[1]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcZ = baseFunctions[idxO[2]];

                    val += dx[neigh] * value(funcX, nowVertex.pos.coords[0]) * value(funcY, nowVertex.pos.coords[1]) * value(funcZ, nowVertex.pos.coords[2]);
                }
            }
            if (nowNode < NodeArraySize) nowNode = NodeArray[nowNode].parent;
            else nowNode = SubdivideArray[nowNode - NodeArraySize].parent;
        }
    }
    SubdivideVvalue[idx] = val - isoValue;
}

__global__ void SparseSurfelFusion::device::generateSubdivideVexNums(const EdgeNode* SubdivideEdgeArray, const EasyOctNode* SubdivideArray, const unsigned int SubdivideEdgeArraySize, const unsigned int NodeArraySize, const float* SubdivideVvalue, int* SubdivideVexNums, bool* markValidSubdivedeVexNum)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideEdgeArraySize)	return;
    int owner = SubdivideEdgeArray[idx].ownerNodeIdx - NodeArraySize;   // ��ǰ�ߵ�Owner
    int kind = SubdivideEdgeArray[idx].edgeKind;                        // ��ǰ�ߵ�����
    int index[2];
    index[0] = edgeVertex[kind][0];
    index[1] = edgeVertex[kind][1];
    int v1 = SubdivideArray[owner].vertices[index[0]] - 1;
    int v2 = SubdivideArray[owner].vertices[index[1]] - 1;
    if (SubdivideVvalue[v1] * SubdivideVvalue[v2] <= 0) {
        SubdivideVexNums[idx] = 1;
        markValidSubdivedeVexNum[idx] = true;
    }
    else {
        markValidSubdivedeVexNum[idx] = false;
    }

}

__global__ void SparseSurfelFusion::device::generateTriNums(const EasyOctNode* SubdivideNodeArray, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, const float* vvalue, int* triNums, int* cubeCatagory)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;
    int currentCubeCatagory = 0;
    for (int i = 0; i < 8; i++) {
        if (vvalue[SubdivideNodeArray[offset].vertices[i] - 1] < 0) {
            currentCubeCatagory |= 1 << i;
        }
    }
    triNums[idx] = trianglesCount[currentCubeCatagory];
    cubeCatagory[idx] = currentCubeCatagory;
}

__global__ void SparseSurfelFusion::device::generateSubdivideIntersectionPoint(const EdgeNode* SubdivideValidEdgeArray, const VertexNode* SubdivideVertexArray, const EasyOctNode* SubdivideArray, const int* SubdivideValidVexAddress, const float* SubdivideVvalue, const unsigned int SubdivideValidEdgeArraySize, const unsigned int NodeArraySize, Point3D<float>* SubdivideVertexBuffer)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideValidEdgeArraySize)	return;
    int owner = SubdivideValidEdgeArray[idx].ownerNodeIdx - NodeArraySize;
    int kind = SubdivideValidEdgeArray[idx].edgeKind;
    int orientation = kind >> 2;
    int index[2];

    index[0] = edgeVertex[kind][0];
    index[1] = edgeVertex[kind][1];

    int v1 = SubdivideArray[owner].vertices[index[0]] - 1;
    int v2 = SubdivideArray[owner].vertices[index[1]] - 1;
    Point3D<float> p1 = SubdivideVertexArray[v1].pos, p2 = SubdivideVertexArray[v2].pos;
    float f1 = SubdivideVvalue[v1];
    float f2 = SubdivideVvalue[v2];
    Point3D<float> isoPoint;
    device::interpolatePoint(p1, p2, orientation, f1, f2, isoPoint);
    SubdivideVertexBuffer[SubdivideValidVexAddress[idx]] = isoPoint;
}

__global__ void SparseSurfelFusion::device::initFixedDepthNums(DeviceArrayView<OctNode> SubdivideNode, DeviceArrayView<int> SubdivideDepthBuffer, const unsigned int DepthOffset, const unsigned int DepthNodeCount, int* fixedDepthNums)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DepthNodeCount)	return;
    const unsigned int offset = DepthOffset + idx;
    int nodeNum = 1;
    for (int depth = SubdivideDepthBuffer[offset]; depth <= device::maxDepth; ++depth) {
        fixedDepthNums[(depth - 1) * DepthNodeCount + idx] = nodeNum;
        nodeNum <<= 3;  // ��8
    }

}

__global__ void SparseSurfelFusion::device::wholeRebuildArray(DeviceArrayView<OctNode> SubdivideNode, const unsigned int finerDepthStart, const unsigned int finerSubdivideNum, const unsigned int NodeArraySize, const int* SubdivideDepthBuffer, const int* depthNodeAddress_Device, const int* fixedDepthAddress, EasyOctNode* RebuildArray, int* RebuildDepthBuffer, Point3D<float>* RebuildCenterBuffer, int* ReplaceNodeId, int* IsRoot, OctNode* NodeArray)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= finerSubdivideNum)	return;
    int depthNodeAddress[MAX_DEPTH_OCTREE + 1];
    for (int i = 0; i <= maxDepth; i++) {                   // ������ݸ��죺L1����
        depthNodeAddress[i] = depthNodeAddress_Device[i];   // depthNodeAddress_Device��Global�ڴ棬������Ҫѭ����ʱ�俪����
    }
    const unsigned int offset = finerDepthStart + idx;
    int nowDepth = SubdivideDepthBuffer[offset];
    int fixedDepthOffset = fixedDepthAddress[(nowDepth - 1) * finerSubdivideNum + idx];
    int nowIdx = depthNodeAddress[nowDepth] + fixedDepthOffset;
    //if (offset % 1000 == 0) printf("idx = %d   nowIdx = %d  nowDepth = %d\n", offset, nowIdx, nowDepth);
    OctNode rootNode = SubdivideNode[offset];
    int replacedId = rootNode.neighs[13];
    rootNode.neighs[13] = NodeArraySize + nowIdx;
    RebuildArray[nowIdx] = rootNode;

    ReplaceNodeId[nowIdx] = replacedId;

    RebuildDepthBuffer[nowIdx] = nowDepth;

    IsRoot[nowIdx] = 1;
    Point3D<float> thisNodeCenter;
    getNodeCenterAllDepth(rootNode.key, nowDepth, thisNodeCenter);
    RebuildCenterBuffer[nowIdx] = thisNodeCenter;
    //if (offset % 1000 == 0) printf("idx = %d   thisNodeCenter = (%.5f, %.5f, %.5f)\n", offset, thisNodeCenter.coords[0], thisNodeCenter.coords[1], thisNodeCenter.coords[2]);

    int sonKey = (rootNode.key >> (3 * (device::maxDepth - nowDepth))) & 7;
    NodeArray[rootNode.parent].children[sonKey] = NodeArraySize + nowIdx;
    int parentNodeIdx;
    int childrenNums = 8;
    while (nowDepth < device::maxDepth) {
        nowDepth++;
        fixedDepthOffset = fixedDepthAddress[(nowDepth - 1) * finerSubdivideNum + idx];
        nowIdx = depthNodeAddress[nowDepth] + fixedDepthOffset;
        for (int j = 0; j < childrenNums; j += 8) {
            int fatherFixedDepthOffset = fixedDepthAddress[(nowDepth - 2) * finerSubdivideNum + idx];
            parentNodeIdx = depthNodeAddress[nowDepth - 1] + fatherFixedDepthOffset + j / 8;
            int parentGlobalIdx = RebuildArray[parentNodeIdx].neighs[13];
            int parentKey = RebuildArray[parentNodeIdx].key;
            for (int k = 0; k < 8; k++) {
                int thisRoundIdx = nowIdx + j + k;
                int nowKey = parentKey | (k << (3 * (device::maxDepth - nowDepth)));
                RebuildArray[thisRoundIdx].parent = parentGlobalIdx;
                RebuildArray[thisRoundIdx].key = nowKey;
                RebuildArray[thisRoundIdx].neighs[13] = NodeArraySize + thisRoundIdx;

                ReplaceNodeId[thisRoundIdx] = replacedId;

                RebuildDepthBuffer[thisRoundIdx] = nowDepth;

                getNodeCenterAllDepth(nowKey, nowDepth, thisNodeCenter);
                RebuildCenterBuffer[thisRoundIdx] = thisNodeCenter;

                RebuildArray[parentNodeIdx].children[k] = NodeArraySize + thisRoundIdx;
            }
        }
        childrenNums <<= 3;
    }
}

__global__ void SparseSurfelFusion::device::markValidMeshVertexIndex(const Point3D<float>* VertexBuffer, const unsigned int verticesNum, bool* markValidVertices)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= verticesNum) return;
    if (fabsf(VertexBuffer[idx].coords[0]) < device::eps) {
        //printf("���� %d ����������", idx);
        markValidVertices[idx] = false;
    }
    else {
        markValidVertices[idx] = true;
    }
}

__global__ void SparseSurfelFusion::device::markValidMeshTriangleIndex(const TriangleIndex* TriangleBuffer, const unsigned int allTriNums, const unsigned int verticesNum, bool* markValidTriangleIndex)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= allTriNums) return;
    bool triValid = true;
    for (int i = 0; i < 3; i++) {
        if (TriangleBuffer[idx].idx[i] < 0 || TriangleBuffer[idx].idx[i] >= verticesNum) {
            triValid = false;
            //printf("�������� %d ���� %d ������ֵ��������!\n", idx, i);
            break;
        }
    }
    if (triValid) {
        markValidTriangleIndex[idx] = true;
    }
    else {
        markValidTriangleIndex[idx] = false;
    }
}



void SparseSurfelFusion::ComputeTriangleIndices::ComputeVertexImplicitFunctionValue(DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> BaseFunction, DeviceArrayView<float> dx, DeviceArrayView<int> encodeNodeIndexInFunction, const float isoValue, hipStream_t stream)
{
    const unsigned int VertexArraySize = VertexArray.Size();
    dim3 block(128);
    dim3 grid(divUp(VertexArraySize, block.x));

    device::ComputeVertexImplicitFunctionValueKernel << <grid, block, 0, stream >> > (VertexArray, NodeArray, BaseFunction, dx, encodeNodeIndexInFunction, VertexArraySize, isoValue, vvalue.Array().ptr());
}

void SparseSurfelFusion::ComputeTriangleIndices::insertTriangle(const Point3D<float>* VertexBufferHost, const int& allVexNums, const int* TriangleBufferHost, const int& allTriNums, CoredVectorMeshData& mesh)
{
    int previousVertex = mesh.inCorePoints.size();

    for (int i = 0; i < allVexNums; i++) {
        if (abs(VertexBufferHost[i].coords[0]) < EPSILON) {
            printf("error\n");
        }
        mesh.inCorePoints.push_back(VertexBufferHost[i]);
    }

    int inCoreFlag = 0; // �ж��ǵڼ����ڵ㣬�����λ�����˳��
    for (int i = 0; i < 3; i++) {
        inCoreFlag |= CoredMeshData::IN_CORE_FLAG[i];
    }

    for (int i = 0; i < allTriNums; i++) {
        TriangleIndex tri;
        for (int j = 0; j < 3; j++) {
            tri.idx[j] = TriangleBufferHost[3 * i + j] + previousVertex;
            //if (i % 100 == 0) printf("depth = %d   idx = %d   TriangleBufferHost[%d] = %d\n" ,depth, i, 3 * i + j, TriangleBufferHost[3 * i + j]);
            if (tri.idx[j] < 0 || tri.idx[j] >= allVexNums + previousVertex) {
                printf("%d %d\n", tri.idx[j] - previousVertex, allVexNums);
                printf("tri error\n");
            }
        }
        mesh.addTriangle(tri, inCoreFlag);
    }
}

void SparseSurfelFusion::ComputeTriangleIndices::insertTriangle(const Point3D<float>* VertexBuffer, const int allVexNums, const TriangleIndex* TriangleBuffer, const int allTriNums, hipStream_t stream)
{
    dim3 block_vex(128);
    dim3 grid_vex(divUp(allVexNums, block_vex.x));
    device::markValidMeshVertexIndex << <grid_vex, block_vex, 0, stream >> > (VertexBuffer, allVexNums, markValidTriangleVertex.Ptr());

    unsigned int* validVerticesCount = NULL;    // ��Ч�Ķ���
    unsigned int validVerticesCountHost = 0;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validVerticesCount), sizeof(unsigned int), stream));

    void* d_temp_storage_1 = NULL;    // �м���������꼴���ͷ�
    size_t temp_storage_bytes_1 = 0;  // �м����
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, VertexBuffer, markValidTriangleVertex.Ptr(), MeshTriangleVertex.Ptr() + MeshTriangleVertex.ArraySize(), validVerticesCount, allVexNums, stream, false));	// ȷ����ʱ�豸�洢����
    CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, VertexBuffer, markValidTriangleVertex.Ptr(), MeshTriangleVertex.Ptr() + MeshTriangleVertex.ArraySize(), validVerticesCount, allVexNums, stream, false));	// ɸѡ	
    CHECKCUDA(hipMemcpyAsync(&validVerticesCountHost, validVerticesCount, sizeof(unsigned int), hipMemcpyDeviceToHost, stream));
    
    dim3 block_tri(128);
    dim3 grid_tri(divUp(allTriNums, block_tri.x));
    device::markValidMeshTriangleIndex << <grid_tri, block_tri, 0, stream >> > (TriangleBuffer, allTriNums, allVexNums, markValidTriangleIndex.Ptr());

    unsigned int* validTriangleIndicesCount = NULL;    // ��Ч������������������
    unsigned int validTriangleIndicesCountHost = 0;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validTriangleIndicesCount), sizeof(unsigned int), stream));

    void* d_temp_storage_2 = NULL;    // �м���������꼴���ͷ�
    size_t temp_storage_bytes_2 = 0;  // �м����
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, TriangleBuffer, markValidTriangleIndex.Ptr(), MeshTriangleIndex.Ptr() + MeshTriangleIndex.ArraySize(), validTriangleIndicesCount, allTriNums, stream, false));	// ȷ����ʱ�豸�洢����
    CHECKCUDA(hipMallocAsync(&d_temp_storage_2, temp_storage_bytes_2, stream));
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, TriangleBuffer, markValidTriangleIndex.Ptr(), MeshTriangleIndex.Ptr() + MeshTriangleIndex.ArraySize(), validTriangleIndicesCount, allTriNums, stream, false));	// ɸѡ	
    CHECKCUDA(hipMemcpyAsync(&validTriangleIndicesCountHost, validTriangleIndicesCount, sizeof(unsigned int), hipMemcpyDeviceToHost, stream));

    CHECKCUDA(hipStreamSynchronize(stream));
    MeshTriangleVertex.ResizeArrayOrException(MeshTriangleVertex.ArraySize() + validVerticesCountHost);
    MeshTriangleIndex.ResizeArrayOrException(MeshTriangleIndex.ArraySize() + validTriangleIndicesCountHost);
}

void SparseSurfelFusion::ComputeTriangleIndices::generateSubdivideNodeArrayCountAndAddress(DeviceBufferArray<OctNode>& NodeArray, DeviceArrayView<unsigned int> DepthBuffer, const unsigned int OtherDepthNodeCount, hipStream_t stream)
{
    SubdivideNode.ResizeArrayOrException(OtherDepthNodeCount);
    CHECKCUDA(hipMemsetAsync(SubdivideNode.Array().ptr(), 0, sizeof(OctNode) * OtherDepthNodeCount, stream));

    int* SubdivideNodeNum = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideNodeNum), sizeof(int), stream));

    ////// �����޷�ʹ��hipcub::DeviceSelect::Flagged������API�ᵼ�¹����ڴ��������Ҫ������L1 Cache �� Share Memory�ı���
    //int* SubdivideNodeNumPtr = NULL;
    //CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideNodeNumPtr), sizeof(int), stream));
    //void* d_temp_storage = NULL;
    //size_t temp_storage_bytes = 0; 
    //CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, NodeArray.Array().ptr(), markValidSubdividedNode.Array().ptr(), SubdivideNode.Array().ptr(), SubdivideNodeNumPtr, OtherDepthNodeCount, stream, false));	// ȷ����ʱ�豸�洢����
    //CHECKCUDA(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
    //CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, NodeArray.Array().ptr(), markValidSubdividedNode.Array().ptr(), SubdivideNode.Array().ptr(), SubdivideNodeNumPtr, OtherDepthNodeCount, stream, false));	// ɸѡ
    //CHECKCUDA(hipMemcpyAsync(&SubdivideNodeNumHost, SubdivideNodeNumPtr, sizeof(int), hipMemcpyDeviceToHost, stream));
    //CHECKCUDA(hipStreamSynchronize(stream));

    // thrust::cuda::par.on(stream) -> ڹ��Thrust��ִ�в�������������ʽ
    thrust::device_ptr<OctNode> NodeArray_ptr = thrust::device_pointer_cast<OctNode>(NodeArray.Array().ptr());
    thrust::device_ptr<OctNode> SubdivideNode_ptr = thrust::device_pointer_cast<OctNode>(SubdivideNode.Array().ptr());
    thrust::device_ptr<OctNode> SubdivideNode_end = thrust::copy_if(thrust::cuda::par.on(stream), NodeArray_ptr, NodeArray_ptr + OtherDepthNodeCount, SubdivideNode_ptr, ifSubdivide());
    CHECKCUDA(hipStreamSynchronize(stream));
    SubdivideNodeNumHost = SubdivideNode_end - SubdivideNode_ptr;   // ɸѡ����Ҫϸ�ֽڵ������

    int* SubdivideDepthNum = NULL;  // ��¼��ǰϸ�ֽڵ����ڵڼ��㣬���ڲ�ȷ��ÿһ��ڵ�����������ÿһ��ϸ�ֽڵ㶼����������������
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideDepthNum), sizeof(int) * SubdivideNodeNumHost * (Constants::maxDepth_Host + 1), stream));
    CHECKCUDA(hipMemsetAsync(SubdivideDepthNum, 0, sizeof(int) * SubdivideNodeNumHost * (Constants::maxDepth_Host + 1), stream));

    SubdivideDepthBuffer.ResizeArrayOrException(SubdivideNodeNumHost);
    
    dim3 block(128);
    dim3 grid(divUp(SubdivideNodeNumHost, block.x));
    device::precomputeSubdivideDepth << <grid, block, 0, stream >> > (SubdivideNode.ArrayView(), DepthBuffer, SubdivideNodeNumHost, SubdivideDepthBuffer.DeviceArray().ptr(), SubdivideDepthNum);

    int* subdivideDepthCount = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&subdivideDepthCount), sizeof(int), stream));


    for (int i = 0; i <= Constants::maxDepth_Host; i++) {
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, SubdivideDepthNum + i * SubdivideNodeNumHost, subdivideDepthCount, SubdivideNodeNumHost, stream);
        CHECKCUDA(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, SubdivideDepthNum + i * SubdivideNodeNumHost, subdivideDepthCount, SubdivideNodeNumHost, stream);
        CHECKCUDA(hipMemcpyAsync(&(SubdivideDepthCount[i]), subdivideDepthCount, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage, stream));
    }

    CHECKCUDA(hipFreeAsync(SubdivideDepthNum, stream));    // ���꼴���ͷţ�����̫��
    CHECKCUDA(hipStreamSynchronize(stream));

    for (int i = 0; i <= Constants::maxDepth_Host; i++) {
        printf("�� %d ��ϸ�ֽڵ�������%d   ", i, SubdivideDepthCount[i]);
        if (i == 0) SubdivideDepthAddress[i] = 0;
        else SubdivideDepthAddress[i] = SubdivideDepthAddress[i - 1] + SubdivideDepthCount[i - 1];
        printf("�ڵ�ƫ�ƣ�%d\n", SubdivideDepthAddress[i]);
    }
}



void SparseSurfelFusion::ComputeTriangleIndices::generateVertexNumsAndVertexAddress(DeviceArrayView<EdgeNode> EdgeArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> vvalue, const unsigned int DLevelOffset, hipStream_t stream)
{
    const unsigned int EdgeArraySize = EdgeArray.Size();
    vexNums.ResizeArrayOrException(EdgeArraySize);
    vexAddress.ResizeArrayOrException(EdgeArraySize);
    markValidVertex.ResizeArrayOrException(EdgeArraySize);
    dim3 block(128);
    dim3 grid(divUp(EdgeArraySize, block.x));
    device::generateVertexNumsKernel << <grid, block, 0, stream >> > (EdgeArray, NodeArray, vvalue, EdgeArraySize, vexNums.Array().ptr(), markValidVertex.Array().ptr());

    void* tempStorage = NULL;	//���㷨��ʱ���������꼴�ͷš�����ǰ׺�͵���ʱ����
    size_t tempStorageBytes = 0;
    hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, vexNums.Array().ptr(), vexAddress.Array().ptr(), EdgeArraySize, stream);
    CHECKCUDA(hipMallocAsync(&tempStorage, tempStorageBytes, stream));
    hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, vexNums.Array().ptr(), vexAddress.Array().ptr(), EdgeArraySize, stream);

    CHECKCUDA(hipFreeAsync(tempStorage, stream));
}

void SparseSurfelFusion::ComputeTriangleIndices::generateTriangleNumsAndTriangleAddress(DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> vvalue, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, hipStream_t stream)
{
    triNums.ResizeArrayOrException(DLevelNodeCount);
    cubeCatagory.ResizeArrayOrException(DLevelNodeCount);
    triAddress.ResizeArrayOrException(DLevelNodeCount);
    dim3 block(128);
    dim3 grid(divUp(DLevelNodeCount, block.x));
    device::generateTriangleNumsKernel << <grid, block, 0, stream >> > (NodeArray, vvalue, DLevelOffset, DLevelNodeCount, triNums.Array().ptr(), cubeCatagory.Array().ptr());

    void* tempStorage = NULL;	//���㷨��ʱ���������꼴�ͷš�����ǰ׺�͵���ʱ����
    size_t tempStorageBytes = 0;
    hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, triNums.Array().ptr(), triAddress.Array().ptr(), DLevelNodeCount, stream);
    CHECKCUDA(hipMallocAsync(&tempStorage, tempStorageBytes, stream));
    hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, triNums.Array().ptr(), triAddress.Array().ptr(), DLevelNodeCount, stream);

    CHECKCUDA(hipFreeAsync(tempStorage, stream));
}

void SparseSurfelFusion::ComputeTriangleIndices::generateVerticesAndTriangle(DeviceBufferArray<OctNode>& NodeArray, DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<EdgeNode> EdgeArray, DeviceArrayView<FaceNode> FaceArray, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, hipStream_t stream)
{
    const unsigned int EdgeArraySize = EdgeArray.Size();
    const unsigned int FaceArraySize = FaceArray.Size();
    printf("VertexSize = %d   EdgeSize = %d   FaceSize = %d\n", VertexArray.Size(), EdgeArraySize, FaceArraySize);
    int lastVexAddr;    // vexAddress�����һ��Ԫ�ر�ʾ���ǣ����������ཻ�ߵ�ƫ�ƣ��ڳ������һ���ߵ�����⣬ǰ���������ཻ�ߵ�����(����exclusiveSum�������ԣ����һ����δͳ��)
    int lastVexNums;    // ���һ�����Ƿ����ؽ������ཻ
    CHECKCUDA(hipMemcpyAsync(&lastVexAddr, vexAddress.Array().ptr() + EdgeArraySize - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECKCUDA(hipMemcpyAsync(&lastVexNums, vexNums.Array().ptr() + EdgeArraySize - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECKCUDA(hipStreamSynchronize(stream));   // ������Ҫͳ��һ�����ж��������

    int allVexNums = lastVexAddr + lastVexNums; // vertex������

    Point3D<float>* VertexBuffer = NULL;    // ��¼�����ؽ������ཻ�Ľ���
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&VertexBuffer), sizeof(Point3D<float>) * allVexNums, stream));

    EdgeNode* validEdgeArray = NULL;        // ��¼��Ч�ı�(���ؽ������ཻ�ı�)����
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validEdgeArray), sizeof(EdgeNode) * allVexNums, stream));

    int* validVertexAddress = NULL;         // ��Ч�ı�(���ؽ������ཻ�ı�)�������е�ƫ�ƣ���������Ч�������index��Ӧ
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validVertexAddress), sizeof(int) * allVexNums, stream));

    int* validEdgeArrayNum = NULL;          // ��¼�ܹ��ж������ؽ������ཻ�ı�
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validEdgeArrayNum), sizeof(int), stream));

    int* validVertexAddressNum = NULL;      // ��¼�ж�����Ч�Ľ���(�����ؽ�����Ľ���)
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validVertexAddressNum), sizeof(int), stream));

    void* d_temp_storage_1 = NULL;
    size_t temp_storage_bytes_1 = 0;
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, EdgeArray.RawPtr(), markValidVertex.Array().ptr(), validEdgeArray, validEdgeArrayNum, EdgeArraySize, stream, false));	// ȷ����ʱ�豸�洢����
    CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, EdgeArray.RawPtr(), markValidVertex.Array().ptr(), validEdgeArray, validEdgeArrayNum, EdgeArraySize, stream, false));	// ɸѡ	

    void* d_temp_storage_2 = NULL;
    size_t temp_storage_bytes_2 = 0;
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, vexAddress.Array().ptr(), markValidVertex.Array().ptr(), validVertexAddress, validVertexAddressNum, EdgeArraySize, stream, false));
    CHECKCUDA(hipMallocAsync(&d_temp_storage_2, temp_storage_bytes_2, stream));
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, vexAddress.Array().ptr(), markValidVertex.Array().ptr(), validVertexAddress, validVertexAddressNum, EdgeArraySize, stream, false));

    assert(allVexNums == validEdgeArrayNum);        // �����Լ��
    assert(allVexNums == validVertexAddressNum);    // �����Լ��

    dim3 block_1(128);
    dim3 grid_1(divUp(allVexNums, block_1.x));
    device::generateIntersectionPoint << <grid_1, block_1, 0, stream >> > (NodeArray.ArrayView(), VertexArray, vvalue.ArrayView(), validEdgeArray, validVertexAddress, allVexNums, VertexBuffer);

    int lastTriAddr;    // ���һ��triAddress��Ԫ�أ������һ���������������������е�ƫ�ƣ��ڳ�ȥ���һ��cube�������������⣬����cube�����ɵ�����������(exclusiveSum��������)
    int lastTriNums;    // ���һ��cube���ɵ������ε�����
    CHECKCUDA(hipMemcpyAsync(&lastTriAddr, triAddress.Array().ptr() + DLevelNodeCount - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECKCUDA(hipMemcpyAsync(&lastTriNums, triNums.Array().ptr() + DLevelNodeCount - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECKCUDA(hipStreamSynchronize(stream));   // ������Ҫͳ��һ�����ж��������
    int allTriNums = lastTriAddr + lastTriNums;

    TriangleIndex* TriangleBuffer = NULL; // ��¼���������ε�����
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&TriangleBuffer), sizeof(TriangleIndex) * allTriNums, stream));

    int* hasSurfaceIntersection = NULL;     // ��¼�Ƿ��������ཻ��cube���ϴ��������ε�һ���߻�����
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&hasSurfaceIntersection), sizeof(int) * FaceArraySize, stream));
    CHECKCUDA(hipMemsetAsync(hasSurfaceIntersection, 0, sizeof(int) * FaceArraySize, stream));

    dim3 block_2(128);
    dim3 grid_2(divUp(DLevelNodeCount, block_2.x));
    device::generateTrianglePos << <grid_2, block_2, 0, stream >> > (NodeArray.ArrayView(), FaceArray, triNums.ArrayView(), cubeCatagory.ArrayView(), vexAddress.ArrayView(), triAddress.ArrayView(), DLevelOffset, DLevelNodeCount, TriangleBuffer, hasSurfaceIntersection);
    CHECKCUDA(hipStreamSynchronize(stream));   // ������Ҫͳ��һ�����ж��������
    printf("�������� = %d   ���������� = %d\n", allVexNums, allTriNums);
    insertTriangle(VertexBuffer, allVexNums, TriangleBuffer, allTriNums, stream);

    //std::vector<Point3D<float>> VertexBufferHost;
    //VertexBufferHost.resize(allVexNums);
    //CHECKCUDA(hipMemcpyAsync(VertexBufferHost.data(), VertexBuffer, sizeof(Point3D<float>) * allVexNums, hipMemcpyDeviceToHost, stream));

    //std::vector<int> TriangleBufferHost;
    //TriangleBufferHost.resize(allTriNums * 3);
    //CHECKCUDA(hipMemcpyAsync(TriangleBufferHost.data(), TriangleBuffer, sizeof(int) * allTriNums * 3, hipMemcpyDeviceToHost, stream));

    

    //insertTriangle(VertexBufferHost.data(), allVexNums, TriangleBufferHost.data(), allTriNums, mesh);

    markValidSubdividedNode.ResizeArrayOrException(DLevelOffset);

    dim3 block_3(128);
    dim3 grid_3(divUp(DLevelOffset, block_3.x));
    device::ProcessLeafNodesAtOtherDepth << <grid_3, block_3, 0, stream >> > (VertexArray, vvalue.ArrayView(), DLevelOffset, hasSurfaceIntersection, NodeArray.Array().ptr(), markValidSubdividedNode.Array().ptr());

    // �������꼴�ͷ�
    CHECKCUDA(hipFreeAsync(VertexBuffer, stream));
    CHECKCUDA(hipFreeAsync(validEdgeArray, stream));
    CHECKCUDA(hipFreeAsync(validVertexAddress, stream));
    CHECKCUDA(hipFreeAsync(validEdgeArrayNum, stream));
    CHECKCUDA(hipFreeAsync(validVertexAddressNum, stream));
    CHECKCUDA(hipFreeAsync(d_temp_storage_1, stream));
    CHECKCUDA(hipFreeAsync(d_temp_storage_2, stream));
    CHECKCUDA(hipFreeAsync(TriangleBuffer, stream));
    CHECKCUDA(hipFreeAsync(hasSurfaceIntersection, stream));

}


void SparseSurfelFusion::ComputeTriangleIndices::CoarserSubdivideNodeAndRebuildMesh(DeviceBufferArray<OctNode>& NodeArray, DeviceArrayView<unsigned int> DepthBuffer, DeviceArrayView<Point3D<float>> CenterBuffer, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> BaseFunction, DeviceArrayView<float> dx, DeviceArrayView<int> encodeNodeIndexInFunction, const float isoValue, hipStream_t stream)
{
    int minSubdivideRootDepth;      // Ѱ�����ϲ�ĸ��ڵ㣬���ڵ������������ýڵ�û�к���(��û�б�ϸ�ֹ�)�����ҽڵ�cube�д��������λ�������ཻ����������ڵ�ϸ��
    SubdivideDepthBuffer.SynchronizeToHost(stream);
    std::vector<int>& SubdivideDepthBufferHost = SubdivideDepthBuffer.HostArray();
    std::vector<OctNode> SubdivideNodeHost;
    SubdivideNode.ArrayView().Download(SubdivideNodeHost);
    minSubdivideRootDepth = SubdivideDepthBufferHost[0];

    //printf("minSubdivideRootDepth = %d\n", minSubdivideRootDepth);

    // ��������Ҫ�����ϲ�����ϸ�������ĸ��ڵ㻮�֣�ȫ�����̵Ļ��֣��ȱ��������
    int maxNodeNums = (powf(8, (Constants::maxDepth_Host - minSubdivideRootDepth + 1)) - 1) / 7;

    //printf("MaxNodesNum = %d\n", maxNodeNums);

    EasyOctNode* SubdivideArray = NULL;     // �ռ�����Coarser Depth�ڵ��У���Ҫ���������ε�ϸ�ֽڵ�
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideArray), sizeof(EasyOctNode) * maxNodeNums, stream));

    int* SubdivideArrayDepthBuffer = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideArrayDepthBuffer), sizeof(int) * maxNodeNums, stream));

    Point3D<float>* SubdivideArrayCenterBuffer = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideArrayCenterBuffer), sizeof(Point3D<float>) * maxNodeNums, stream));

    for (int i = 0; i < SubdivideNodeNumHost; i++) {
        int rootDepth = SubdivideDepthBufferHost[i];
        if (rootDepth >= finerDepth) break; // ֻ����ϸ�ֵ�root�ڵ���� < finerDepth��
        // ��rootDepth����ڵ�Ϊ�����ڵ�(�൱��0��ڵ�)���������е�Nodeȫ�����ţ���ȫ�ָ���СΪ��a0(1 - q^n)/(1-q) = 1(1 - 8^(maxDepth - rootDepth + 1))/(1 - 8)
        int SubdivideArraySize = (powf(8, (Constants::maxDepth_Host - rootDepth + 1)) - 1) / 7;
        int currentNodeNum = 1;
        for (int j = rootDepth; j <= Constants::maxDepth_Host; j++) {
            fixedDepthNodeNum[j] = currentNodeNum;
            currentNodeNum <<= 3;       // ��8
        }

        //for (int j = 0; j <= Constants::maxDepth_Host; j++) {
        //    printf("fixedDepthNodeNum[%d] = %d\n", j, fixedDepthNodeNum[j]);
        //}

        //printf("SubdivideArraySize - (D-1) = %d   fixedDepthNodeNum = %d\n", SubdivideArraySize - fixedDepthNodeNum[Constants::maxDepth_Host - 1], fixedDepthNodeNum[Constants::maxDepth_Host]);

        for (int j = rootDepth; j <= Constants::maxDepth_Host; j++) {
            fixedDepthNodeAddress[j] = fixedDepthNodeAddress[j - 1] + fixedDepthNodeNum[j - 1];
        }
        OctNode rootNode = SubdivideNodeHost[i];    // ��ǰ����Ƚڵ�
        int rootIndex = rootNode.neighs[13];        // ��ǰ����Ƚڵ��NodeArray����
        int rootParent = rootNode.parent;           // ��ǰ����Ƚڵ�ĸ��ڵ�
        int rootKey = rootNode.key;                 // ��ǰ����Ƚڵ��Key
        int rootSonKey = (rootKey >> (3 * (Constants::maxDepth_Host - rootDepth))) & 7; // ��ǰ����Ƚڵ��ڵ�ǰ�����λ����

        // �����һ���ڵ������
        CHECKCUDA(hipMemsetAsync(SubdivideArray, 0, sizeof(EasyOctNode) * SubdivideArraySize, stream));

        int NodeArraySize = NodeArray.ArraySize();
        OctNode* NodeArrayPtr = NodeArray.Array().ptr();
        // ��ϸ�ֺ����ĵ��index������NodeArray���棬index��NodeArraySize��ʼ���涼��ϸ�ֵĵ�
        CHECKCUDA(hipMemcpyAsync(&NodeArrayPtr[rootParent].children[rootSonKey], &NodeArraySize, sizeof(int), hipMemcpyHostToDevice, stream));
        // �����ڵ�ĸ��ף���ΪSubdivideArray��ʼԪ�صĸ��ף�SubdivideArray[0]���ǵ�ǰϸ���µ�root(0��)
        CHECKCUDA(hipMemcpyAsync(&SubdivideArray[0].parent, &rootParent, sizeof(int), hipMemcpyHostToDevice, stream));

        dim3 block_1(128);
        dim3 grid_1(divUp(SubdivideArraySize, block_1.x));
        device::singleRebuildArray << <grid_1, block_1, 0, stream >> > (SubdivideNode.ArrayView(), SubdivideDepthBuffer.DeviceArrayReadOnly(), i, NodeArraySize, SubdivideArraySize, SubdivideArray, SubdivideArrayDepthBuffer, SubdivideArrayCenterBuffer);

        for (int depth = rootDepth; depth <= Constants::maxDepth_Host; depth++) {
            dim3 block_2(128);
            dim3 grid_2(divUp(fixedDepthNodeNum[depth], block_2.x));
            device::computeRebuildNeighbor << <grid_2, block_2, 0, stream >> > (NodeArray.ArrayView(), fixedDepthNodeAddress[depth], fixedDepthNodeNum[depth], NodeArraySize, depth, SubdivideArray);
        }

        /**************************************** SubdivideVertexArray ****************************************/

        VertexNode* SubdividePreVertexArray = NULL;     // ��ʱ���������꼴ɾ
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdividePreVertexArray), sizeof(VertexNode) * 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdividePreVertexArray, 0, sizeof(VertexNode) * 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        markValidSubdivideVertex.ResizeArrayOrException(fixedDepthNodeNum[Constants::maxDepth_Host]);

        dim3 block_3(128);
        dim3 grid_3(divUp(fixedDepthNodeNum[Constants::maxDepth_Host], block_3.x));
        device::initSubdivideVertexOwner << <grid_3, block_3, 0, stream >> > (SubdivideArray, SubdivideArrayCenterBuffer, fixedDepthNodeAddress[Constants::maxDepth_Host], fixedDepthNodeNum[Constants::maxDepth_Host], NodeArraySize, SubdividePreVertexArray, markValidSubdivideVertex.Array().ptr());
    
        VertexNode* SubdivideVertexArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVertexArray), sizeof(VertexNode) * 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideVertexArray, 0, sizeof(VertexNode) * 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
    
        int* SubdivideVertexArraySize = NULL;
        int SubdivideVertexArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVertexArraySize), sizeof(int), stream));

        void* d_temp_storage_1 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_1 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, SubdividePreVertexArray, markValidSubdivideVertex.Array().ptr(), SubdivideVertexArray, SubdivideVertexArraySize, 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, SubdividePreVertexArray, markValidSubdivideVertex.Array().ptr(), SubdivideVertexArray, SubdivideVertexArraySize, 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&SubdivideVertexArraySizeHost, SubdivideVertexArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(SubdividePreVertexArray, stream));  // ��ʱ���������꼴ɾ
        CHECKCUDA(hipFreeAsync(SubdivideVertexArraySize, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage_1, stream));

        CHECKCUDA(hipStreamSynchronize(stream));       // ͬ���������SubdivideVertexArraySizeHost

        if (SubdivideVertexArraySizeHost == 0) {        // �ýڵ�û�п�ϸ�ֵĶ��㡾����ʽ���棬����ͱ�ȱһ���ɡ�
            CHECKCUDA(hipFreeAsync(SubdivideVertexArray, stream));
            continue;
        }
        //printf("SubdivideVertexArraySizeHost = %d\n", SubdivideVertexArraySizeHost);

        dim3 block_4(128);
        dim3 grid_4(divUp(SubdivideVertexArraySizeHost, block_4.x));
        device::maintainSubdivideVertexNodePointer << <grid_4, block_4, 0, stream >> > (CenterBuffer, SubdivideVertexArraySizeHost, NodeArraySize, SubdivideArrayCenterBuffer, SubdivideVertexArray, SubdivideArray);

        /**************************************** SubdivideEdgeArray ****************************************/

        EdgeNode* SubdividePreEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdividePreEdgeArray), sizeof(EdgeNode) * 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdividePreEdgeArray, 0, sizeof(EdgeNode) * 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        markValidSubdivideEdge.ResizeArrayOrException(fixedDepthNodeNum[Constants::maxDepth_Host]);

        dim3 block_5(128);
        dim3 grid_5(divUp(fixedDepthNodeNum[Constants::maxDepth_Host], block_5.x));
        device::initSubdivideEdgeArray << <grid_5, block_5, 0, stream >> > (SubdivideArray, SubdivideArrayCenterBuffer, NodeArraySize, fixedDepthNodeAddress[Constants::maxDepth_Host], fixedDepthNodeNum[Constants::maxDepth_Host], SubdividePreEdgeArray, markValidSubdivideEdge.Array().ptr());

        EdgeNode* SubdivideEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideEdgeArray), sizeof(EdgeNode) * 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideEdgeArray, 0, sizeof(EdgeNode) * 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        int* SubdivideEdgeArraySize = NULL;
        int SubdivideEdgeArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideEdgeArraySize), sizeof(int), stream));

        void* d_temp_storage_2 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_2 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, SubdividePreEdgeArray, markValidSubdivideEdge.Array().ptr(), SubdivideEdgeArray, SubdivideEdgeArraySize, 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_2, temp_storage_bytes_2, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, SubdividePreEdgeArray, markValidSubdivideEdge.Array().ptr(), SubdivideEdgeArray, SubdivideEdgeArraySize, 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&SubdivideEdgeArraySizeHost, SubdivideEdgeArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(SubdividePreEdgeArray, stream));  // ��ʱ���������꼴ɾ
        CHECKCUDA(hipFreeAsync(SubdivideEdgeArraySize, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage_2, stream));

        CHECKCUDA(hipStreamSynchronize(stream));       // ͬ���������SubdivideEdgeArraySizeHost

        if (SubdivideEdgeArraySizeHost == 0) {          // �ýڵ�û�п�ϸ�ֵıߡ�����ʽ���棬����ͱ�ȱһ���ɡ�
            CHECKCUDA(hipFreeAsync(SubdivideVertexArray, stream));
            CHECKCUDA(hipFreeAsync(SubdivideEdgeArray, stream));
            continue;
        }

        //printf("SubdivideEdgeArraySizeHost = %d\n", SubdivideEdgeArraySizeHost);
        

        dim3 block_6(128);
        dim3 grid_6(divUp(SubdivideEdgeArraySizeHost, block_6.x));
        device::maintainSubdivideEdgeNodePointer << <grid_6, block_6, 0, stream >> > (CenterBuffer, SubdivideArrayCenterBuffer, SubdivideEdgeArraySizeHost, NodeArraySize, SubdivideArray, SubdivideEdgeArray);

        /**************************************** ����ϸ�ֽڵ���ʽ������ֵ, ����ϸ�ֶ����vexNums��vexAddress ****************************************/
        float* SubdivideVvalue = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVvalue), sizeof(float) * SubdivideVertexArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(SubdivideVvalue, 0, sizeof(float) * SubdivideVertexArraySizeHost, stream));
        dim3 block_7(128);
        dim3 grid_7(divUp(SubdivideVertexArraySizeHost, block_7.x));
        device::computeSubdivideVertexImplicitFunctionValue << <grid_7, block_7, 0, stream >> > (SubdivideVertexArray, SubdivideArray, NodeArray.ArrayView(), dx, encodeNodeIndexInFunction, BaseFunction, NodeArraySize, rootIndex, SubdivideVertexArraySizeHost, isoValue, SubdivideVvalue);

        int* SubdivideVexNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVexNums), sizeof(int) * SubdivideEdgeArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(SubdivideVexNums, 0, sizeof(int) * SubdivideEdgeArraySizeHost, stream));

        markValidSubdivedeVexNum.ResizeArrayOrException(SubdivideEdgeArraySizeHost);
        
        dim3 block_8(128);
        dim3 grid_8(divUp(SubdivideEdgeArraySizeHost, block_8.x));
        device::generateSubdivideVexNums << <grid_8, block_8, 0, stream >> > (SubdivideEdgeArray, SubdivideArray, SubdivideEdgeArraySizeHost, NodeArraySize, SubdivideVvalue, SubdivideVexNums, markValidSubdivedeVexNum.Array().ptr());
      
        int* SubdivideVexAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVexAddress), sizeof(int) * SubdivideEdgeArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(SubdivideVexAddress, 0, sizeof(int) * SubdivideEdgeArraySizeHost, stream));

        void* tempVexAddressStorage = NULL;	//���㷨��ʱ���������꼴�ͷš�����ǰ׺�͵���ʱ����
        size_t tempVexAddressStorageBytes = 0;
        hipcub::DeviceScan::ExclusiveSum(tempVexAddressStorage, tempVexAddressStorageBytes, SubdivideVexNums, SubdivideVexAddress, SubdivideEdgeArraySizeHost, stream);
        CHECKCUDA(hipMallocAsync(&tempVexAddressStorage, tempVexAddressStorageBytes, stream));
        hipcub::DeviceScan::ExclusiveSum(tempVexAddressStorage, tempVexAddressStorageBytes, SubdivideVexNums, SubdivideVexAddress, SubdivideEdgeArraySizeHost, stream);

        CHECKCUDA(hipFreeAsync(tempVexAddressStorage, stream));

        int SubdivideLastVexAddr = -1;
        int SubdivideLastVexNums = -1;
        CHECKCUDA(hipMemcpyAsync(&SubdivideLastVexAddr, SubdivideVexAddress + SubdivideEdgeArraySizeHost - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipMemcpyAsync(&SubdivideLastVexNums, SubdivideVexNums + SubdivideEdgeArraySizeHost - 1, sizeof(int), hipMemcpyDeviceToHost, stream));


        CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��

        int SubdivideAllVexNums = SubdivideLastVexAddr + SubdivideLastVexNums;
        //printf("SubdivideAllVexNums = %d\n", SubdivideAllVexNums);

        if (SubdivideAllVexNums == 0) {
            CHECKCUDA(hipMemcpyAsync(&(NodeArray[rootParent].children[rootSonKey]), &rootIndex, sizeof(int), hipMemcpyHostToDevice, stream));
            CHECKCUDA(hipFreeAsync(SubdivideVertexArray, stream));
            CHECKCUDA(hipFreeAsync(SubdivideEdgeArray, stream));
            CHECKCUDA(hipFreeAsync(SubdivideVvalue, stream));
            CHECKCUDA(hipFreeAsync(SubdivideVexNums, stream));
            CHECKCUDA(hipFreeAsync(SubdivideVexAddress, stream));
            continue;
        }

        /**************************************** ����ϸ�ֶ���������κ����������� ****************************************/

        int* SubdivideTriNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideTriNums), sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideTriNums, 0, sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        int* SubdivideCubeCatagory = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideCubeCatagory), sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideCubeCatagory, 0, sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        dim3 block_9(128);
        dim3 grid_9(divUp(fixedDepthNodeNum[Constants::maxDepth_Host], block_9.x));
        device::generateTriNums << <grid_9, block_9, 0, stream >> > (SubdivideArray, fixedDepthNodeAddress[Constants::maxDepth_Host], fixedDepthNodeNum[Constants::maxDepth_Host], SubdivideVvalue, SubdivideTriNums, SubdivideCubeCatagory);

        int* SubdivideTriAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideTriAddress), sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideTriAddress, 0, sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        void* tempTriAddressStorage = NULL;	//���㷨��ʱ���������꼴�ͷš�����ǰ׺�͵���ʱ����
        size_t tempTriAddressStorageBytes = 0;
        hipcub::DeviceScan::ExclusiveSum(tempTriAddressStorage, tempTriAddressStorageBytes, SubdivideTriNums, SubdivideTriAddress, fixedDepthNodeNum[Constants::maxDepth_Host], stream);
        CHECKCUDA(hipMallocAsync(&tempTriAddressStorage, tempTriAddressStorageBytes, stream));
        hipcub::DeviceScan::ExclusiveSum(tempTriAddressStorage, tempTriAddressStorageBytes, SubdivideTriNums, SubdivideTriAddress, fixedDepthNodeNum[Constants::maxDepth_Host], stream);

        CHECKCUDA(hipFreeAsync(tempTriAddressStorage, stream));

        Point3D<float>* SubdivideVertexBuffer = NULL;
        //std::vector<Point3D<float>> SubdivideVertexBufferHost;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVertexBuffer), sizeof(Point3D<float>) * SubdivideAllVexNums, stream));
        //SubdivideVertexBufferHost.resize(SubdivideAllVexNums);

        EdgeNode* SubdivideValidEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideValidEdgeArray), sizeof(EdgeNode) * SubdivideAllVexNums, stream));

        int* SubdivideValidEdgeArraySize = NULL;    // ��Ч��ϸ�ֱ�device
        int SubdivideValidEdgeArraySizeHost = -1;   // ��Ч��ϸ�ֱ�Host
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideValidEdgeArraySize), sizeof(int), stream));

        void* d_temp_storage_3 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_3 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_3, temp_storage_bytes_3, SubdivideEdgeArray, markValidSubdivedeVexNum.Array().ptr(), SubdivideValidEdgeArray, SubdivideValidEdgeArraySize, SubdivideEdgeArraySizeHost, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_3, temp_storage_bytes_3, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_3, temp_storage_bytes_3, SubdivideEdgeArray, markValidSubdivedeVexNum.Array().ptr(), SubdivideValidEdgeArray, SubdivideValidEdgeArraySize, SubdivideEdgeArraySizeHost, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&SubdivideValidEdgeArraySizeHost, SubdivideValidEdgeArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        int* SubdivideValidVexAddress = NULL;    // ��Ч��ϸ�ֱ�device
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideValidVexAddress), sizeof(int) * SubdivideAllVexNums, stream));

        int* SubdivideValidVexAddressSize = NULL;    // ��Ч��ϸ�ֱ�device
        int SubdivideValidVexAddressSizeHost = -1;   // ��Ч��ϸ�ֱ�Host
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideValidVexAddressSize), sizeof(int), stream));

        void* d_temp_storage_4 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_4 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_4, temp_storage_bytes_4, SubdivideVexAddress, markValidSubdivedeVexNum.Array().ptr(), SubdivideValidVexAddress, SubdivideValidVexAddressSize, SubdivideEdgeArraySizeHost, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_4, temp_storage_bytes_4, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_4, temp_storage_bytes_4, SubdivideVexAddress, markValidSubdivedeVexNum.Array().ptr(), SubdivideValidVexAddress, SubdivideValidVexAddressSize, SubdivideEdgeArraySizeHost, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&SubdivideValidVexAddressSizeHost, SubdivideValidVexAddressSize, sizeof(int), hipMemcpyDeviceToHost, stream));

        dim3 block_10(128);
        dim3 grid_10(divUp(SubdivideAllVexNums, block_10.x));
        device::generateSubdivideIntersectionPoint << <grid_10, block_10, 0, stream >> > (SubdivideValidEdgeArray, SubdivideVertexArray, SubdivideArray, SubdivideValidVexAddress, SubdivideVvalue, SubdivideValidEdgeArraySizeHost, NodeArraySize, SubdivideVertexBuffer);
        //CHECKCUDA(hipMemcpyAsync(SubdivideVertexBufferHost.data(), SubdivideVertexBuffer, sizeof(Point3D<float>) * SubdivideAllVexNums, hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(SubdivideValidEdgeArray, stream));
        CHECKCUDA(hipFreeAsync(SubdivideValidVexAddress, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage_3, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage_4, stream));

        //CHECKCUDA(hipStreamSynchronize(stream));
        //printf("depth = %d   SubdivideValidVexAddressSize = %d\n", i, SubdivideValidVexAddressSizeHost);

        int SubdivideLastTriAddr;
        int SubdivideLastTriNums;
        CHECKCUDA(hipMemcpyAsync(&SubdivideLastTriAddr, SubdivideTriAddress + fixedDepthNodeNum[Constants::maxDepth_Host] - 1, sizeof(int), hipMemcpyDeviceToHost));
        CHECKCUDA(hipMemcpyAsync(&SubdivideLastTriNums, SubdivideTriNums + fixedDepthNodeNum[Constants::maxDepth_Host] - 1, sizeof(int), hipMemcpyDeviceToHost));
        CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        int SubdivideAllTriNums = SubdivideLastTriAddr + SubdivideLastTriNums;
        //printf("depth = %d   SubdivideAllTriNums = %d\n", i, SubdivideAllTriNums);

        TriangleIndex* SubdivideTriangleBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideTriangleBuffer), sizeof(TriangleIndex) * SubdivideAllTriNums, stream));
        //std::vector<int> SubdivideTriangleBufferHost;
        //SubdivideTriangleBufferHost.resize(3 * SubdivideAllTriNums);

        dim3 block_11(128);
        dim3 grid_11(divUp(fixedDepthNodeNum[Constants::maxDepth_Host], block_11.x));
        device::generateSubdivideTrianglePos << <grid_11, block_11, 0, stream >> > (SubdivideArray, fixedDepthNodeAddress[Constants::maxDepth_Host], fixedDepthNodeNum[Constants::maxDepth_Host], SubdivideTriNums, SubdivideCubeCatagory, SubdivideVexAddress, SubdivideTriAddress, SubdivideTriangleBuffer);
        insertTriangle(SubdivideVertexBuffer, SubdivideAllVexNums, SubdivideTriangleBuffer, SubdivideAllTriNums, stream);
        //CHECKCUDA(hipMemcpyAsync(SubdivideTriangleBufferHost.data(), SubdivideTriangleBuffer, sizeof(int) * 3 * SubdivideAllTriNums, hipMemcpyDeviceToHost, stream));

        //CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        //
        //insertTriangle(SubdivideVertexBufferHost.data(), SubdivideAllVexNums, SubdivideTriangleBufferHost.data(), SubdivideAllTriNums, mesh);

        CHECKCUDA(hipMemcpy(&(NodeArray[rootParent].children[rootSonKey]), &rootIndex, sizeof(int), hipMemcpyHostToDevice));
        CHECKCUDA(hipFreeAsync(SubdivideVertexArray, stream));
        CHECKCUDA(hipFreeAsync(SubdivideEdgeArray, stream));
        CHECKCUDA(hipFreeAsync(SubdivideVvalue, stream));
        CHECKCUDA(hipFreeAsync(SubdivideVexNums, stream));
        CHECKCUDA(hipFreeAsync(SubdivideVexAddress, stream));
        CHECKCUDA(hipFreeAsync(SubdivideTriNums, stream));
        CHECKCUDA(hipFreeAsync(SubdivideCubeCatagory, stream));
        CHECKCUDA(hipFreeAsync(SubdivideTriAddress, stream));
        CHECKCUDA(hipFreeAsync(SubdivideVertexBuffer, stream));
        CHECKCUDA(hipFreeAsync(SubdivideTriangleBuffer, stream));
    }
    CHECKCUDA(hipFreeAsync(SubdivideArray, stream));
    CHECKCUDA(hipFreeAsync(SubdivideArrayCenterBuffer, stream));
    CHECKCUDA(hipFreeAsync(SubdivideArrayDepthBuffer, stream));
}

void SparseSurfelFusion::ComputeTriangleIndices::FinerSubdivideNodeAndRebuildMesh(DeviceBufferArray<OctNode>& NodeArray, DeviceArrayView<unsigned int> DepthBuffer, DeviceArrayView<Point3D<float>> CenterBuffer, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> BaseFunction, DeviceArrayView<float> dx, DeviceArrayView<int> encodeNodeIndexInFunction, const float isoValue, hipStream_t stream)
{
    const unsigned int NodeArraySize = NodeArray.ArraySize();
    for (int i = finerDepth; i < Constants::maxDepth_Host; i++) {
        int finerDepthStart = SubdivideDepthAddress[i];
        int finerSubdivideNum = SubdivideDepthCount[i];
        int* fixedDepthNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&fixedDepthNums), sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, stream));
        CHECKCUDA(hipMemsetAsync(fixedDepthNums, 0, sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, stream));

        dim3 block_1(128);
        dim3 grid_1(divUp(finerSubdivideNum, block_1.x));
        device::initFixedDepthNums << <grid_1, block_1, 0, stream >> > (SubdivideNode.ArrayView(), SubdivideDepthBuffer.DeviceArrayReadOnly(), finerDepthStart, finerSubdivideNum, fixedDepthNums);

        int* rebuildNumsDevice = NULL;
        int rebuildNums = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&rebuildNumsDevice), sizeof(int), stream));
        
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, fixedDepthNums, rebuildNumsDevice, finerSubdivideNum * Constants::maxDepth_Host, stream);
        CHECKCUDA(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, fixedDepthNums, rebuildNumsDevice, finerSubdivideNum * Constants::maxDepth_Host, stream);
        CHECKCUDA(hipMemcpyAsync(&rebuildNums, rebuildNumsDevice, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage, stream));
        CHECKCUDA(hipFreeAsync(rebuildNumsDevice, stream));

        for (int depth = 1; depth <= Constants::maxDepth_Host; depth++) {
            void* d_temp_storage_1 = NULL;
            size_t temp_storage_bytes_1 = 0;
            int* LevelNodeCount = NULL;
            CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&LevelNodeCount), sizeof(int), stream));
            hipcub::DeviceReduce::Sum(d_temp_storage_1, temp_storage_bytes_1, fixedDepthNums + (depth - 1) * finerSubdivideNum, LevelNodeCount, finerSubdivideNum, stream);
            CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
            hipcub::DeviceReduce::Sum(d_temp_storage_1, temp_storage_bytes_1, fixedDepthNums + (depth - 1) * finerSubdivideNum, LevelNodeCount, finerSubdivideNum, stream);
            CHECKCUDA(hipMemcpyAsync(&depthNodeCount[depth], LevelNodeCount, sizeof(int), hipMemcpyDeviceToHost, stream));
            CHECKCUDA(hipFreeAsync(d_temp_storage_1, stream));
            CHECKCUDA(hipFreeAsync(LevelNodeCount, stream));
        }

        for (int depth = 0; depth <= Constants::maxDepth_Host; depth++) {
            if (depth == 0) depthNodeAddress[depth] = 0;
            else {
                depthNodeAddress[depth] = depthNodeAddress[depth - 1] + depthNodeCount[depth - 1];
            }
        }

        int* depthNodeAddress_Device = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&depthNodeAddress_Device), sizeof(int) * (Constants::maxDepth_Host + 1), stream));
        CHECKCUDA(hipMemcpyAsync(depthNodeAddress_Device, depthNodeAddress, sizeof(int) * (Constants::maxDepth_Host + 1), hipMemcpyHostToDevice, stream));

        int* fixedDepthAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&fixedDepthAddress), sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, stream));
        CHECKCUDA(hipMemsetAsync(fixedDepthAddress, 0, sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, stream));
        for (int depth = 1; depth <= Constants::maxDepth_Host; depth++) {
            void* d_temp_storage = NULL;
            size_t temp_storage_bytes = 0;
            hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, fixedDepthNums + (depth - 1) * finerSubdivideNum, fixedDepthAddress + (depth - 1) * finerSubdivideNum, finerSubdivideNum, stream);
            CHECKCUDA(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
            hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, fixedDepthNums + (depth - 1) * finerSubdivideNum, fixedDepthAddress + (depth - 1) * finerSubdivideNum, finerSubdivideNum, stream);
            CHECKCUDA(hipFreeAsync(d_temp_storage, stream));
        }

        CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ�������rebuildNums

        const unsigned int rebuildDLevelCount = rebuildNums - depthNodeAddress[Constants::maxDepth_Host];

        EasyOctNode* RebuildArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildArray), sizeof(EasyOctNode) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(RebuildArray, 0, sizeof(EasyOctNode) * rebuildNums, stream));

        int* RebuildDepthBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildDepthBuffer), sizeof(int) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(RebuildDepthBuffer, 0, sizeof(int) * rebuildNums, stream));

        Point3D<float>* RebuildCenterBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildCenterBuffer), sizeof(Point3D<float>) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(RebuildCenterBuffer, 0, sizeof(Point3D<float>) * rebuildNums, stream));

        int* ReplaceNodeId = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&ReplaceNodeId), sizeof(int) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(ReplaceNodeId, 0, sizeof(int) * rebuildNums, stream));

        int* IsRoot = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&IsRoot), sizeof(int) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(IsRoot, 0, sizeof(int) * rebuildNums, stream));

        dim3 block_2(128);
        dim3 grid_2(divUp(finerSubdivideNum, block_2.x));
        device::wholeRebuildArray << <grid_2, block_2, 0, stream >> > (SubdivideNode.ArrayView(), finerDepthStart, finerSubdivideNum, NodeArraySize, SubdivideDepthBuffer.DeviceArray().ptr(), depthNodeAddress_Device, fixedDepthAddress, RebuildArray, RebuildDepthBuffer, RebuildCenterBuffer, ReplaceNodeId, IsRoot, NodeArray.Array().ptr());
        
        for (int depth = finerDepth; depth <= Constants::maxDepth_Host; depth++) {
            dim3 block(128);
            dim3 grid(divUp(depthNodeCount[depth], block.x));
            device::computeRebuildNeighbor << <grid, block, 0, stream >> > (NodeArray.ArrayView(), depthNodeAddress[depth], depthNodeCount[depth], NodeArraySize, depth, RebuildArray);
        }

        VertexNode* RebuildPreVertexArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildPreVertexArray), sizeof(VertexNode) * rebuildDLevelCount * 8, stream));
        CHECKCUDA(hipMemsetAsync(RebuildPreVertexArray, 0, sizeof(VertexNode) * rebuildDLevelCount * 8, stream));

        markValidFinerVexArray.ResizeArrayOrException(rebuildDLevelCount * 8);

        dim3 block_3(128);
        dim3 grid_3(divUp(rebuildDLevelCount, block_3.x));
        device::initSubdivideVertexOwner << <grid_3, block_3, 0, stream >> > (RebuildArray, RebuildCenterBuffer, depthNodeAddress[Constants::maxDepth_Host], rebuildDLevelCount, NodeArraySize, RebuildPreVertexArray, markValidFinerVexArray.Array().ptr());
        
        VertexNode* RebuildVertexArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVertexArray), sizeof(VertexNode) * rebuildDLevelCount * 8, stream));
        CHECKCUDA(hipMemsetAsync(RebuildVertexArray, 0, sizeof(VertexNode) * rebuildDLevelCount * 8, stream));

        int* RebuildVertexArraySize = NULL;
        int RebuildVertexArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVertexArraySize), sizeof(int), stream));

        void* d_temp_storage_1 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_1 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, RebuildPreVertexArray, markValidFinerVexArray.Array().ptr(), RebuildVertexArray, RebuildVertexArraySize, 8 * rebuildDLevelCount, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, RebuildPreVertexArray, markValidFinerVexArray.Array().ptr(), RebuildVertexArray, RebuildVertexArraySize, 8 * rebuildDLevelCount, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&RebuildVertexArraySizeHost, RebuildVertexArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(RebuildPreVertexArray, stream));   // ��ʱ��������ʱɾ��
        CHECKCUDA(hipFreeAsync(RebuildVertexArraySize, stream));  // ��ʱ��������ʱɾ��
        CHECKCUDA(hipFreeAsync(d_temp_storage_1, stream));        // ��ʱ��������ʱɾ��

        CHECKCUDA(hipStreamSynchronize(stream));
        if (RebuildVertexArraySizeHost == 0) {                      // ϸ�ֶ���Ϊ0����ֱ����һ�㡾������ʽ����ֵ������ͱ�ȱһ���ɡ�
            CHECKCUDA(hipFreeAsync(fixedDepthNums, stream));
            CHECKCUDA(hipFreeAsync(depthNodeAddress_Device, stream));
            CHECKCUDA(hipFreeAsync(fixedDepthAddress, stream));
            CHECKCUDA(hipFreeAsync(RebuildArray, stream));
            CHECKCUDA(hipFreeAsync(RebuildDepthBuffer, stream));
            CHECKCUDA(hipFreeAsync(RebuildCenterBuffer, stream));
            CHECKCUDA(hipFreeAsync(ReplaceNodeId, stream));
            CHECKCUDA(hipFreeAsync(IsRoot, stream));
            CHECKCUDA(hipFreeAsync(RebuildVertexArray, stream));
            continue;
        }

        dim3 block_4(128);
        dim3 grid_4(divUp(RebuildVertexArraySizeHost, block_4.x));
        device::maintainSubdivideVertexNodePointer << <grid_4, block_4, 0, stream >> > (CenterBuffer, RebuildVertexArraySizeHost, NodeArraySize, RebuildCenterBuffer, RebuildVertexArray, RebuildArray);

        EdgeNode* RebuildPreEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildPreEdgeArray), sizeof(EdgeNode) * rebuildDLevelCount * 12, stream));
        CHECKCUDA(hipMemsetAsync(RebuildPreEdgeArray, 0, sizeof(EdgeNode) * rebuildDLevelCount * 12, stream));

        markValidFinerEdge.ResizeArrayOrException(rebuildDLevelCount * 12);

        dim3 block_5(128);
        dim3 grid_5(divUp(rebuildDLevelCount, block_5.x));
        device::initSubdivideEdgeArray << <grid_5, block_5, 0, stream >> > (RebuildArray, RebuildCenterBuffer, NodeArraySize, depthNodeAddress[Constants::maxDepth_Host], rebuildDLevelCount, RebuildPreEdgeArray, markValidFinerEdge.Array().ptr());

        EdgeNode* RebuildEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildEdgeArray), sizeof(EdgeNode)* rebuildDLevelCount * 12, stream));
        CHECKCUDA(hipMemsetAsync(RebuildEdgeArray, 0, sizeof(EdgeNode)* rebuildDLevelCount * 12, stream));

        int* RebuildEdgeArraySize = NULL;
        int RebuildEdgeArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildEdgeArraySize), sizeof(int), stream));

        void* d_temp_storage_2 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_2 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, RebuildPreEdgeArray, markValidFinerEdge.Array().ptr(), RebuildEdgeArray, RebuildEdgeArraySize, 12 * rebuildDLevelCount, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_2, temp_storage_bytes_2, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, RebuildPreEdgeArray, markValidFinerEdge.Array().ptr(), RebuildEdgeArray, RebuildEdgeArraySize, 12 * rebuildDLevelCount, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&RebuildEdgeArraySizeHost, RebuildEdgeArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(RebuildPreEdgeArray, stream));      // ��ʱ��������ʱɾ��
        CHECKCUDA(hipFreeAsync(RebuildEdgeArraySize, stream));     // ��ʱ��������ʱɾ��
        CHECKCUDA(hipFreeAsync(d_temp_storage_2, stream));         // ��ʱ��������ʱɾ��

        CHECKCUDA(hipStreamSynchronize(stream));   // ������Ҫͬ��
        if (RebuildEdgeArraySizeHost == 0) {                      // ϸ�ֱ�Ϊ0����ֱ����һ�㡾������ʽ����ֵ������ͱ�ȱһ���ɡ�
            CHECKCUDA(hipFreeAsync(fixedDepthNums, stream));
            CHECKCUDA(hipFreeAsync(depthNodeAddress_Device, stream));
            CHECKCUDA(hipFreeAsync(fixedDepthAddress, stream));
            CHECKCUDA(hipFreeAsync(RebuildArray, stream));
            CHECKCUDA(hipFreeAsync(RebuildDepthBuffer, stream));
            CHECKCUDA(hipFreeAsync(RebuildCenterBuffer, stream));
            CHECKCUDA(hipFreeAsync(ReplaceNodeId, stream));
            CHECKCUDA(hipFreeAsync(IsRoot, stream));
            CHECKCUDA(hipFreeAsync(RebuildVertexArray, stream));
            CHECKCUDA(hipFreeAsync(RebuildEdgeArray, stream));
            continue;
        }

        //printf("depth = %d  RebuildVertexArraySize = %d  RebuildEdgeArraySize = %d\n", i, RebuildVertexArraySizeHost, RebuildEdgeArraySizeHost);

        dim3 block_6(128);
        dim3 grid_6(divUp(RebuildEdgeArraySizeHost, block_6.x));
        device::maintainSubdivideEdgeNodePointer << <grid_6, block_6, 0, stream >> > (CenterBuffer, RebuildCenterBuffer, RebuildEdgeArraySizeHost, NodeArraySize, RebuildArray, RebuildEdgeArray);

        float* RebuildVvalue = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVvalue), sizeof(float) * RebuildVertexArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(RebuildVvalue, 0, sizeof(float) * RebuildVertexArraySizeHost, stream));

        dim3 block_7(128);
        dim3 grid_7(divUp(RebuildVertexArraySizeHost, block_7.x));
        device::computeSubdivideVertexImplicitFunctionValue << <grid_7, block_7, 0, stream >> > (RebuildVertexArray, RebuildArray, NodeArray.ArrayView(), dx, encodeNodeIndexInFunction, BaseFunction, NodeArraySize, ReplaceNodeId, IsRoot, RebuildVertexArraySizeHost, isoValue, RebuildVvalue);

        CHECKCUDA(hipFreeAsync(ReplaceNodeId, stream));
        CHECKCUDA(hipFreeAsync(IsRoot, stream));

        int* RebuildVexNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVexNums), sizeof(int) * RebuildEdgeArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(RebuildVexNums, 0, sizeof(int) * RebuildEdgeArraySizeHost, stream));

        dim3 block_8(128);
        dim3 grid_8(divUp(RebuildEdgeArraySizeHost, block_8.x));
        device::generateSubdivideVexNums << <grid_8, block_8, 0, stream >> > (RebuildEdgeArray, RebuildArray, RebuildEdgeArraySizeHost, NodeArraySize, RebuildVvalue, RebuildVexNums, markValidFinerVexNum.Array().ptr());

        int* RebuildVexAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVexAddress), sizeof(int) * RebuildEdgeArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(RebuildVexAddress, 0, sizeof(int) * RebuildEdgeArraySizeHost, stream));

        void* d_temp_storage_3 = NULL;
        size_t temp_storage_bytes_3 = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage_3, temp_storage_bytes_3, RebuildVexNums, RebuildVexAddress, RebuildEdgeArraySizeHost, stream);
        CHECKCUDA(hipMallocAsync(&d_temp_storage_3, temp_storage_bytes_3, stream));
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage_3, temp_storage_bytes_3, RebuildVexNums, RebuildVexAddress, RebuildEdgeArraySizeHost, stream);
        CHECKCUDA(hipFreeAsync(d_temp_storage_3, stream));

        CHECKCUDA(hipStreamSynchronize(stream));   // ��Ҫͬ��
        int RebuildLastVexAddr = -1;
        int RebuildLastVexNums = -1;
        CHECKCUDA(hipMemcpyAsync(&RebuildLastVexAddr, RebuildVexAddress + RebuildEdgeArraySizeHost - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipMemcpyAsync(&RebuildLastVexNums, RebuildVexNums + RebuildEdgeArraySizeHost - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
        int RebuildAllVexNums = RebuildLastVexAddr + RebuildLastVexNums;

        CHECKCUDA(hipStreamSynchronize(stream));   // ��Ҫͬ��
        //printf("depth = %d   RebuildAllVexNums = %d\n", i, RebuildAllVexNums);

        int* RebuildTriNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildTriNums), sizeof(int) * rebuildDLevelCount, stream));
        CHECKCUDA(hipMemsetAsync(RebuildTriNums, 0, sizeof(int) * rebuildDLevelCount, stream));

        int* RebuildCubeCatagory = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildCubeCatagory), sizeof(int) * rebuildDLevelCount, stream));
        CHECKCUDA(hipMemsetAsync(RebuildCubeCatagory, 0, sizeof(int) * rebuildDLevelCount, stream));

        dim3 block_9(128);
        dim3 grid_9(divUp(rebuildDLevelCount, block_9.x));
        device::generateTriNums << <grid_9, block_9, 0, stream >> > (RebuildArray, depthNodeAddress[Constants::maxDepth_Host], rebuildDLevelCount, RebuildVvalue, RebuildTriNums, RebuildCubeCatagory);

        int* RebuildTriAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildTriAddress), sizeof(int)* rebuildDLevelCount, stream));
        CHECKCUDA(hipMemsetAsync(RebuildTriAddress, 0, sizeof(int)* rebuildDLevelCount, stream));

        void* d_temp_storage_4 = NULL;
        size_t temp_storage_bytes_4 = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage_4, temp_storage_bytes_4, RebuildTriNums, RebuildTriAddress, rebuildDLevelCount, stream);
        CHECKCUDA(hipMallocAsync(&d_temp_storage_4, temp_storage_bytes_4, stream));
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage_4, temp_storage_bytes_4, RebuildTriNums, RebuildTriAddress, rebuildDLevelCount, stream);
        CHECKCUDA(hipFreeAsync(d_temp_storage_4, stream));

        Point3D<float>* RebuildVertexBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVertexBuffer), sizeof(Point3D<float>)* RebuildAllVexNums, stream));
        //std::vector<Point3D<float>> RebuildVertexBufferHost;
        //RebuildVertexBufferHost.resize(RebuildAllVexNums);

        EdgeNode* RebuildValidEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildValidEdgeArray), sizeof(EdgeNode)* RebuildAllVexNums, stream));

        int* RebuildValidEdgeArraySize = NULL;
        int RebuildValidEdgeArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildValidEdgeArraySize), sizeof(int), stream));

        void* d_temp_storage_5 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_5 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_5, temp_storage_bytes_5, RebuildEdgeArray, markValidFinerVexNum.Array().ptr(), RebuildValidEdgeArray, RebuildValidEdgeArraySize, RebuildEdgeArraySizeHost, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_5, temp_storage_bytes_5, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_5, temp_storage_bytes_5, RebuildEdgeArray, markValidFinerVexNum.Array().ptr(), RebuildValidEdgeArray, RebuildValidEdgeArraySize, RebuildEdgeArraySizeHost, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&RebuildValidEdgeArraySizeHost, RebuildValidEdgeArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(d_temp_storage_5, stream));
        CHECKCUDA(hipFreeAsync(RebuildValidEdgeArraySize, stream));

        int* RebuildValidVexAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildValidVexAddress), sizeof(int)* RebuildAllVexNums, stream));

        int* RebuildValidVexAddressSize = NULL;
        int RebuildValidVexAddressSizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildValidVexAddressSize), sizeof(int), stream));

        void* d_temp_storage_6 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_6 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_6, temp_storage_bytes_6, RebuildVexAddress, markValidFinerVexNum.Array().ptr(), RebuildValidVexAddress, RebuildValidVexAddressSize, RebuildEdgeArraySizeHost, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_6, temp_storage_bytes_6, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_6, temp_storage_bytes_6, RebuildVexAddress, markValidFinerVexNum.Array().ptr(), RebuildValidVexAddress, RebuildValidVexAddressSize, RebuildEdgeArraySizeHost, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&RebuildValidVexAddressSizeHost, RebuildValidVexAddressSize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(d_temp_storage_6, stream));
        CHECKCUDA(hipFreeAsync(RebuildValidVexAddressSize, stream));

        dim3 block_10(128);
        dim3 grid_10(divUp(RebuildAllVexNums, block_10.x));
        device::generateSubdivideIntersectionPoint << <grid_10, block_10, 0, stream >> > (RebuildValidEdgeArray, RebuildVertexArray, RebuildArray, RebuildValidVexAddress, RebuildVvalue, RebuildAllVexNums, NodeArraySize, RebuildVertexBuffer);

        CHECKCUDA(hipFreeAsync(RebuildValidEdgeArray, stream));
        CHECKCUDA(hipFreeAsync(RebuildValidVexAddress, stream));
        CHECKCUDA(hipFreeAsync(RebuildVvalue, stream));

        int RebuildLastTriAddr = -1;
        int RebuildLastTriNums = -1;

        CHECKCUDA(hipMemcpyAsync(&RebuildLastTriAddr, RebuildTriAddress + rebuildDLevelCount - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipMemcpyAsync(&RebuildLastTriNums, RebuildTriNums + rebuildDLevelCount - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    
        CHECKCUDA(hipStreamSynchronize(stream));
        int RebuildAllTriNums = RebuildLastTriAddr + RebuildLastTriNums;

        CHECKCUDA(hipFreeAsync(RebuildVexNums, stream));

        //printf("RebuildAllTriNums = %d\n", RebuildAllTriNums);

        TriangleIndex* RebuildTriangleBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildTriangleBuffer), sizeof(TriangleIndex) * RebuildAllTriNums, stream));
        //std::vector<int> RebuildTriangleBufferHost;
        //RebuildTriangleBufferHost.resize(RebuildAllTriNums * 3);        

        CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        //printf("###############################   Depth = %d   #################################\n", i);

        dim3 block_11(128);
        dim3 grid_11(divUp(rebuildDLevelCount, block_11.x));
        device::generateSubdivideTrianglePos << <grid_11, block_11, 0, stream >> > (RebuildArray, depthNodeAddress[Constants::maxDepth_Host], rebuildDLevelCount, RebuildTriNums, RebuildCubeCatagory, RebuildVexAddress, RebuildTriAddress, RebuildTriangleBuffer);
        insertTriangle(RebuildVertexBuffer, RebuildAllVexNums, RebuildTriangleBuffer, RebuildAllTriNums, stream);
        //CHECKCUDA(hipMemcpyAsync(RebuildVertexBufferHost.data(), RebuildVertexBuffer, sizeof(Point3D<float>) * RebuildAllVexNums, hipMemcpyDeviceToHost, stream));
        //CHECKCUDA(hipMemcpyAsync(RebuildTriangleBufferHost.data(), RebuildTriangleBuffer, sizeof(int) * 3 * RebuildAllTriNums, hipMemcpyDeviceToHost, stream));

        //CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��

        //insertTriangle(RebuildVertexBufferHost.data(), RebuildAllVexNums, RebuildTriangleBufferHost.data(), RebuildAllTriNums, mesh);

        CHECKCUDA(hipFreeAsync(fixedDepthNums, stream));
        CHECKCUDA(hipFreeAsync(depthNodeAddress_Device, stream));
        CHECKCUDA(hipFreeAsync(fixedDepthAddress, stream));
        CHECKCUDA(hipFreeAsync(RebuildArray, stream));
        CHECKCUDA(hipFreeAsync(RebuildDepthBuffer, stream));
        CHECKCUDA(hipFreeAsync(RebuildCenterBuffer, stream));
        CHECKCUDA(hipFreeAsync(RebuildVertexArray, stream));
        CHECKCUDA(hipFreeAsync(RebuildEdgeArray, stream));
        CHECKCUDA(hipFreeAsync(RebuildVexAddress, stream));
        CHECKCUDA(hipFreeAsync(RebuildTriNums, stream));
        CHECKCUDA(hipFreeAsync(RebuildCubeCatagory, stream));
        CHECKCUDA(hipFreeAsync(RebuildTriAddress, stream));
        CHECKCUDA(hipFreeAsync(RebuildVertexBuffer, stream));
        CHECKCUDA(hipFreeAsync(RebuildTriangleBuffer, stream));
    }
}
